#include "hip/hip_runtime.h"
#include <ctime>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <builtin_types.h>
#include <crtdbg.h>

#include"Common.h"
#include"GlobalSpace.h"
#include "Timer.h"
#include"sortingNetworks_common.cuh"
#include "MSA-GA-CUDA.h"

namespace SeqAnsis
{

#define	SubMatPos(r,c)	(r)*(r+1)/2+(c)

CMSAGA_CUDA_Algorithm::CMSAGA_CUDA_Algorithm():m_pPopulation(NULL), m_fMaxScore(-INT_MAX), m_iRun(0), m_pSubMatDevice(NULL)
{

}

CMSAGA_CUDA_Algorithm::~CMSAGA_CUDA_Algorithm()
{
	SAFE_DELETE_ARRAY(m_pPopulation);
}

void	CMSAGA_CUDA_Algorithm::SetAlignParams(int nPopulationNum, int nNoAdvGenerationNum, int nMaxOrgLen, const std::vector<float>& SeqWeight)
{
	int n_seq = (int)SeqWeight.size();
	MSA_GA_CUDA_MAX_LEN = nMaxOrgLen;
	R_MSA_GA_CUDA_MAX_LEN = 1.0f / MSA_GA_CUDA_MAX_LEN;
	MSA_GA_POPULATION_SIZE = nPopulationNum;
	MSA_GA_NO_ADV_GENERATION_NUM = nNoAdvGenerationNum;
	POPULATION_WIDTH = MSA_GA_CUDA_MAX_LEN*MSA_GA_POPULATION_SIZE;
	R_POPULATION_WIDTH = 1.0f / POPULATION_WIDTH;
	//Init weight of sequences
	m_SeqWeight.clear();
	m_SeqWeight.resize(SeqWeight.size());

#ifdef WEIGHT_SUM_SP_SCORE
	float sum = 0.0f;
	for (int i = 0; i < (int)SeqWeight.size() - 1; ++i)
	{
		for (int j = i + 1; j < (int)SeqWeight.size(); ++j)
		{
			sum += SeqWeight[i] * SeqWeight[j];
		}
	}
#endif

	float sum = 0.0f;
	for (int i = 0; i < (int)SeqWeight.size(); ++i)
	{
		sum += SeqWeight[i];
	}

	float similarity = 1.0-sum/n_seq;

#ifdef WEIGHT_SUM_SP_SCORE
	for (int i = 0; i < (int)SeqWeight.size(); ++i)
	{
		m_SeqWeight[i] = SeqWeight[i] / sqrt(sum);
	}
#endif

#ifdef CLUSTALW_SP_SCORE
	for (int i = 0; i < (int)SeqWeight.size(); ++i)
	{
		m_SeqWeight[i] = SeqWeight[i];
	}
#else
	for (int i = 0; i < (int)SeqWeight.size(); ++i)
	{
		m_SeqWeight[i] = 1.0f;
	}
#endif

#if 0
	sum = 0.0f;
	for (int i = 0; i < (int)SeqWeight.size() - 1; ++i)
	{
		for (int j = i + 1; j < (int)SeqWeight.size(); ++j)
		{
			sum += m_SeqWeight[i] * m_SeqWeight[j];
		}
	}
	sum += 0;
#endif
}

void CMSAGA_CUDA_Algorithm::Align( std::vector<CSequence>& vAlignedSequences, const std::vector<CSequence>& vSequences )
{
	m_OrginSequences = vSequences;
	int n_seq = m_OrginSequences.size();
	m_rnPairs = 2.0f / (n_seq*(n_seq - 1));
	m_bdbgFirstRun = true;
	//find the  SP score of the original sequences
	int score = -1;
	//score = SPScore(vSequences);
	char ch[100];
	sprintf_s(ch, "The input sequences SP score=%i (-1 means not scoring!)", score);
	CGlobalSpace::m_sEventLog.writeEvent(ch);

	//population initialization
	InitPopulation( vSequences );

	//init all data needed for device computing.
	InitDevice();
	
	//calculate the fitness of every organism. if not meet the requirement ,then evolution begin.
	CGlobalSpace::m_sEventLog.writeEvent( "MSA-GA-CUDA: Population evolution begin!" );
	CTimer  time;
	double dStartTime = time.getCurrentTime();

////============FOR CPU  VERIFICATION================//
//	short*	score = new short[MSA_GA_POPULATION_SIZE];
//	for ( int i=0; i<MSA_GA_POPULATION_SIZE; ++i )
//	{
//		m_pPopulation[i].score = SPScore( m_pPopulation[i] );
//		score[i] = m_pPopulation[i].score;
//	}
//	std::auto_ptr<SeqAnsis::CFileWriter>		pFileWriter( new SeqAnsis::CFileWriter( "cpuScore.txt" ) );
//	pFileWriter->openFile();
//	pFileWriter->OutputVector( score,  MSA_GA_POPULATION_SIZE);
//	pFileWriter->closeFile();
//	SAFE_DELETE_ARRAY(score);
//	
//	FitnessDeviceUnlimit();
//	dbgWriteFile("m_pOrganScore_Device.txt", m_pOrganScore_Device,MSA_GA_POPULATION_SIZE,MSA_GA_POPULATION_SIZE);
////============FOR CPU  VERIFICATION================//

	m_iRun = 0;
	int iCount = 0;
	float maxScore = INT_MIN;
	while(!FitnessDeviceUnlimit() && m_iRun<MSA_GA_GENERATION_NUM )//&& iCount<MSA_GA_NO_ADV_GENERATION_NUM)
	{
		Evolution();
		if (m_bestScore>maxScore)
		{
			iCount = 0;
			maxScore = m_bestScore;
		}
		else
		{
			++iCount;
		}
		++m_iRun;		
		if (m_bdbgFirstRun)
		{
			sprintf_s(ch, "generation=%i, maxscore=%f", 1, maxScore);
			CGlobalSpace::m_sEventLog.writeEvent(ch);
			m_bdbgFirstRun = false;
		}
	}
	double dInterval = time.getCurrentTime() - dStartTime;
	sprintf_s(ch, "the process time of CUDA-MSA-GA algorithm is %f", dInterval);
	SeqAnsis::CGlobalSpace::m_sEventLog.writeEvent(ch);

	//dbgCheckSequence();
	sprintf_s(ch, "generation=%i, maxscore=%f, iCount=%i", m_iRun, maxScore, iCount);
	CGlobalSpace::m_sEventLog.writeEvent(ch);
	dbgWriteSeqsFromDevice2File( "The final align sequences.txt" );
	//dbgWriteFile("OrganScore_Sorted_Device.txt", m_pOrganScore_Align_Sorted_Device, MSA_GA_POPULATION_SIZE, MSA_GA_POPULATION_SIZE);
	CGlobalSpace::m_sEventLog.writeEvent( "MSA-GA-CUDA: Population evolution end!" );
	ReadAlignedSeqsFromDevice( vAlignedSequences );

	ClearDevice();
}

void     CMSAGA_CUDA_Algorithm::ReadAlignedSeqsFromDevice( std::vector<CSequence>& vAlignedSequences )
{
	unsigned int size_organs = m_pPopulation->nSeqSize*MSA_GA_CUDA_MAX_LEN*MSA_GA_POPULATION_SIZE;
	unsigned int mem_size_organs = sizeof(short) * size_organs;
	short *h_organs = (short *)malloc(mem_size_organs);
	//read sequences from device to host
	CUDA_DEV2HOST(   h_organs, m_pPopulationDevice[m_curPopulationIndex], mem_size_organs );

	//fill the alignedSequences with best aligned one
	int nWidth = MSA_GA_CUDA_MAX_LEN*MSA_GA_POPULATION_SIZE;
	for( int i=0; i<m_pPopulation->nSeqSize; ++i )
	{
		int idx = 0;
		struct StruSeqElem  seqElem;
		CSeqData  seqData;
		for ( int j=0; j<MSA_GA_CUDA_MAX_LEN; ++j )
		{
			if( h_organs[ i*nWidth+j ] != MSA_GA_CUDA_SPACE )
			{
				seqElem.m_iCode = h_organs[ i*nWidth+j ];
				seqElem.m_char = CGlobalSpace::m_sAlignParams.getAminoAcidInt2CharCode( seqElem.m_iCode );
				if( seqElem.m_char != GENESPACE || seqElem.m_char!=NONEGENE )
				{
					seqElem.m_index = idx;
					++idx;
				}
				else
				{
					seqElem.m_index = -1;
				}
				seqData.push_back( seqElem );
			}
			else
			{
				break;
			}
		}
		CSequence seq( seqData,  m_pPopulation[0].pSequence[i].sequence.getName(), m_pPopulation[0].pSequence[i].sequence.getTitle() );
		vAlignedSequences.push_back( seq );
	}
	free(h_organs);
	h_organs=NULL;
}

__device__ inline void devMOD(int M, int N, float revN, int& quotient, int& remainder)
{
	quotient = floor(M*revN);
	remainder = M - quotient*N;
}

template <int BLOCK_SIZE> __global__ void
MSAGA_CALC_SEQ_LEN( bool bRecomb, unsigned int *d_OrganLenNextGen, unsigned int *d_OrganLen, short *d_population, float *d_rand_organ_idx, float fChance, int nOrganWidth, int nPopulationNum)
{	
	// Thread index
	int tx = threadIdx.x;

	if ( tx==0 )
	{
		d_OrganLenNextGen[tx] = d_OrganLen[tx];
		return;
	}

	int  curIdx = -1;
	if (bRecomb)
	{
		if ((nPopulationNum - 1) % 2 != 0 && tx == (nPopulationNum - 1))
		{//�ܸ���Ϊ���������һ������ֱ�Ӹ���
			d_OrganLenNextGen[tx] = d_OrganLen[tx];
			return;
		}
		else
		{
			int  curOrder = tx;
			if (tx % 2 == 0)//�����ǰ������ż����Ÿ���
			{
				curOrder = tx - 1;//ʹ����������������Ž��м���
			}
			curIdx = curOrder / 2;//half of the population size
		}
	}
	else
	{
		curIdx = tx;
	}

	if (d_rand_organ_idx[curIdx]<fChance)
	{	
		int len=0;
		int i=0;
		while( d_population[tx*nOrganWidth+i]!=MSA_GA_CUDA_SPACE && i<nOrganWidth )
		{
			++len;
			++i;
		}
		d_OrganLenNextGen[tx]=len;
	}
	else
	{
		d_OrganLenNextGen[tx]=d_OrganLen[tx];
	}
}

template <int BLOCK_SIZE> __global__ void
MSAGA_CALC_SEQ_LEN_UNLIMIT(bool bRecomb, unsigned int *d_OrganLenNextGen, unsigned int *d_OrganLen, short *d_population, float *d_rand_organ_idx, float fChance, int nOrganWidth, int nWidth, int nPopulationNum)
{	
	// Block index
	int bx = blockIdx.x;
	//    int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int orgIdx = bx*blockDim.x + ty*nWidth + tx;
	if (orgIdx >= nPopulationNum)  return;

	if (orgIdx == 0)
	{
		d_OrganLenNextGen[orgIdx] = d_OrganLen[orgIdx];
		return;
	}

	int  curIdx = -1;
	if (bRecomb)
	{
		//result=(MSA_GA_POPULATION_SIZE - 1) % 2
		int j = (nPopulationNum - 1) * 0.5f;
		int result = (nPopulationNum - 1) - j * 2;
		if (result != 0 && orgIdx == (nPopulationNum - 1))
		{//�ܸ���Ϊ���������һ������ֱ�Ӹ���
			d_OrganLenNextGen[orgIdx] = d_OrganLen[orgIdx];
			return;
		}
		else
		{
			int  curOrder = orgIdx;
			//result = orgIdx % 2
			int j = orgIdx * 0.5f;
			int result = orgIdx - j * 2;
			if (result == 0)//�����ǰ������ż����Ÿ���
			{
				curOrder = orgIdx - 1;//ʹ����������������Ž��м���
			}
			curIdx = curOrder / 2;//half of the population size
		}
	}
	else
	{
		curIdx = orgIdx;
	}

	if (d_rand_organ_idx[curIdx]<fChance)
	{	
		int len=0;
		int i=0;
		while (d_population[orgIdx*nOrganWidth + i] != MSA_GA_CUDA_SPACE && i<nOrganWidth)
		{
			++len;
			++i;
		}
		d_OrganLenNextGen[orgIdx] = len;
	}
	else
	{
		d_OrganLenNextGen[orgIdx]=d_OrganLen[orgIdx];
	}
}

#if 0
__device__ inline void ComparatorShort(
    short &keyA,
    short &valA,
    short &keyB,
    short &valB,
    short dir
)
{
    short t;

    if ((keyA > keyB) == dir)
    {
        t = keyA;
        keyA = keyB;
        keyB = t;
        t = valA;
        valA = valB;
        valB = t;
    }
}

////////////////////////////////////////////////////////////////////////////////
// Monolithic bitonic sort kernel for short arrays fitting into shared memory
////////////////////////////////////////////////////////////////////////////////
__global__ void bitonicSortShared(
    short *d_DstKey,
    short *d_DstVal,
    short *d_SrcKey,
    short *d_SrcVal,
    short arrayLength,
    short dir
)
{
    //Shared memory storage for one or more short vectors
    __shared__ short s_key[SHARED_SIZE_LIMIT];
    __shared__ short s_val[SHARED_SIZE_LIMIT];

    //Offset to the beginning of subbatch and load data
    d_SrcKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_SrcVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    d_DstVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
    s_key[threadIdx.x +                       0] = d_SrcKey[                      0];
    s_val[threadIdx.x +                       0] = d_SrcVal[                      0];
    s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcKey[(SHARED_SIZE_LIMIT / 2)];
    s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = d_SrcVal[(SHARED_SIZE_LIMIT / 2)];
	
    for (uint size = 2; size < arrayLength; size <<= 1)
    {
        //Bitonic merge
        uint ddd = dir ^ ((threadIdx.x & (size / 2)) != 0);

        for (uint stride = size / 2; stride > 0; stride >>= 1)
        {
            __syncthreads();
            uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            ComparatorShort(
                s_key[pos +      0], s_val[pos +      0],
                s_key[pos + stride], s_val[pos + stride],
                ddd
            );
        }
    }
	
    //ddd == dir for the last bitonic merge step
    {
        for (uint stride = arrayLength / 2; stride > 0; stride >>= 1)
        {
            __syncthreads();
            uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            ComparatorShort(
                s_key[pos +      0], s_val[pos +      0],
                s_key[pos + stride], s_val[pos + stride],
                dir
            );
        }
    }

    __syncthreads();
    d_DstKey[                      0] = s_key[threadIdx.x +                       0];
    d_DstVal[                      0] = s_val[threadIdx.x +                       0];
    d_DstKey[(SHARED_SIZE_LIMIT / 2)] = s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
    d_DstVal[(SHARED_SIZE_LIMIT / 2)] =  s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
}
#endif

void		CMSAGA_CUDA_Algorithm::InitPopulation( const std::vector<CSequence>& vSequences )
{
	int iLongestSeqSize = -INT_MAX;
	for (int i = 0; i<(int)vSequences.size(); ++i)
	{
		m_iLongestSeqSize = iLongestSeqSize<vSequences[i].getLen() ? vSequences[i].getLen() : iLongestSeqSize;

	}
	if (m_pPopulation)	SAFE_DELETE_ARRAY(m_pPopulation);
	m_pPopulation = new COrganism[MSA_GA_POPULATION_SIZE];
	m_OrganismSize = vSequences.size();
	for ( int i=0; i<MSA_GA_POPULATION_SIZE; ++i )
	{
		m_pPopulation[i].pSequence = new CMSA_GASeq[vSequences.size()];
		m_pPopulation[i].nSeqSize=vSequences.size();
		m_pPopulation[i].score = 0;

		for ( int j=0; j<(int)vSequences.size(); ++j )
		{
			m_pPopulation[i].pSequence[j].charNum = CountSeqCharNum( vSequences[j] );
			m_pPopulation[i].pSequence[j].gapOffset =  CGlobalSpace::m_sUtility.getRandomNumber(MSA_GA_OFFSET_FACTOR_X*m_iLongestSeqSize);
			CSeqData  tmpSeqData;
			for ( int k=0; k<m_pPopulation[i].pSequence[j].gapOffset; ++k )
			{
				tmpSeqData.push_back( StruSeqElem(GENESPACE,-1) );
			}
			for ( int k=0; k<vSequences[j].getLen(); ++k )
			{
				tmpSeqData.push_back( StruSeqElem( vSequences[j].getSequenceContext().at(k).m_char, vSequences[j].getSequenceContext().at(k).m_index ) );
			}
			m_pPopulation[i].pSequence[j].sequence = CSequence( tmpSeqData, vSequences[j].getName(), vSequences[j].getTitle() );
		}
	}
	ArrangeSequences();//��CPU����������

	//load the sequence alignment from HOST to DEVICE
	unsigned int size_organs = m_pPopulation->nSeqSize*MSA_GA_CUDA_MAX_LEN*MSA_GA_POPULATION_SIZE;
	unsigned int mem_size_organs = sizeof(short) * size_organs;
	short *h_organs = (short *)malloc(mem_size_organs);
	unsigned int mem_size_organLen = sizeof(unsigned int)*MSA_GA_POPULATION_SIZE;
	unsigned int *h_organLen = (unsigned int *)malloc(mem_size_organLen);
	int n=0;
	for ( int j=0; j<MSA_GA_POPULATION_SIZE; ++j )
	{
		const CSeqData& seq = m_pPopulation[j].pSequence[0].sequence.getSequenceContext();
		h_organLen[j] = seq.size();
	}

	for( int i=0; i<m_pPopulation->nSeqSize; ++i )
	{
		for ( int j=0; j<MSA_GA_POPULATION_SIZE; ++j )
		{
			const CSeqData& seq = m_pPopulation[j].pSequence[i].sequence.getSequenceContext();
			if( seq.size()>MSA_GA_CUDA_MAX_LEN )
			{
				free(h_organs);
				h_organs  = NULL;
				free(h_organLen);
				h_organLen=NULL;
				throw   CAppException( DEF_EXCEPTION_INDEX_OUT_OF_RANGE,DEF_EXCEPTION_LEVEL_EXIT_TRY_BLOCK
					,__EXCEPTION_SITE__ ,"The max length of the sequence exceed the MSA_GA_CUDA_MAX_LEN" );
			}
			for( int k=0; k<seq.size(); ++k )
			{
				h_organs[n] = short(seq[k].m_iCode);
				++n;
			}
			for( int k=seq.size(); k<MSA_GA_CUDA_MAX_LEN; ++k )
			{
				h_organs[n] = MSA_GA_CUDA_SPACE;
				++n;
			}
		}
	}
	//copy the data to Device
	m_curPopulationIndex = 0;
	CUDA_MALLOC( (void**)&m_pPopulationDevice[m_curPopulationIndex],  h_organs, mem_size_organs );
	CUDA_MALLOC( (void**)&m_pPopulationDevice[1-m_curPopulationIndex],  mem_size_organs );
	m_curOrganLenIndex=0;
	CUDA_MALLOC( (void**)&m_pOrganLenDevice[m_curOrganLenIndex], h_organLen,  mem_size_organLen );
	CUDA_MALLOC( (void**)&m_pOrganLenDevice[1-m_curPopulationIndex],  mem_size_organLen );
	free(h_organs);
	h_organs=NULL;
	free(h_organLen);
	h_organLen=NULL;

	//�������е���Ч���㵥Ԫ�ĸ���
	m_nCellNum = POPULATION_WIDTH*m_pPopulation->nSeqSize;

	//dbgWriteSeqsFromDevice2File("The init align sequences.txt");

	CGlobalSpace::m_sEventLog.writeEvent( "MSA-GA-CUDA: Population Initialized and load to Device!" );
}

int		CMSAGA_CUDA_Algorithm::CountSeqCharNum( const CSequence& vSeq )
{
	int iLen = 0;
	for( int k=0; k<vSeq.getLen(); ++k )
	{
		if ( vSeq.getSequenceContext().at(k).m_iCode!=CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE) )
		{
			++iLen;
		}
	}
	return iLen;
}

void     CMSAGA_CUDA_Algorithm::ResizeOrgan( COrganism& vOrgan )
{
	CSeqData::iterator	*itrSeq(new CSeqData::iterator[vOrgan.nSeqSize]);
	for ( int j=0; j<vOrgan.nSeqSize; ++j )
	{
		itrSeq[j] = vOrgan.pSequence[j].sequence.getSequenceContext().begin();
	}

	int iLongestLength = GetLongestSeqLen( vOrgan );
	for ( int j=0; j<iLongestLength; ++j )
	{
		bool		bSpace = true;
		for ( int k=0; k<vOrgan.nSeqSize; ++k )
		{
			if ( itrSeq[k]!=vOrgan.pSequence[k].sequence.getSequenceContext().end() && itrSeq[k]->m_iCode != CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE) )
			{
				bSpace = false;
				break;
			}
		}
		if( bSpace )	
		{
			for ( int k=0; k<vOrgan.nSeqSize; ++k )
			{
				if ( itrSeq[k]!=vOrgan.pSequence[k].sequence.getSequenceContext().end() )
				{
					itrSeq[k] = vOrgan.pSequence[k].sequence.getSequenceContext().erase( itrSeq[k] );
				}
			}
		}
		else
		{
			for ( int k=0; k<vOrgan.nSeqSize; ++k )
			{
				if ( itrSeq[k]!=vOrgan.pSequence[k].sequence.getSequenceContext().end() )
				{
					++itrSeq[k];
				}
			}
		}
	}

	SAFE_DELETE_ARRAY(itrSeq);

	//recalculate the length after it has changed.
	iLongestLength = GetLongestSeqLen( vOrgan );

	//adding GENSPACE at the shorter sequence tail.
	for ( int j=0; j<vOrgan.nSeqSize; ++j )
	{
		if ( vOrgan.pSequence[j].sequence.getLen()<iLongestLength )
		{
			int orglen = vOrgan.pSequence[j].sequence.getLen();
			for ( int k=0; k<(iLongestLength-orglen); ++k )
			{
				vOrgan.pSequence[j].sequence.getSequenceContext().push_back( StruSeqElem(GENESPACE, -1) );
			}				
		}
	}
}

void		CMSAGA_CUDA_Algorithm::ArrangeSequences()
{
	for ( int i=0; i<MSA_GA_POPULATION_SIZE; ++i )
	{
		ResizeOrgan( m_pPopulation[i] );
	}
}

int		CMSAGA_CUDA_Algorithm::GetLongestSeqLen( const COrganism& og )
{
	//find the length of the longest sequence
	int iLongestLength = og.pSequence[0].sequence.getLen();
	for ( int j=1; j<og.nSeqSize; ++j )
	{
		if ( iLongestLength<og.pSequence[j].sequence.getLen() )
		{
			iLongestLength = og.pSequence[j].sequence.getLen();
		}
	}
	return iLongestLength;
}

__global__ void  MSAGA_SPS_UNIT_CUDA_UNLIMIT(float *d_score, short *d_population, unsigned int *d_populationLen, short *d_subMat, float *d_seqWeight, int nSeq, int nWidth, int nPopulationWidth, float rnPopulationWidth, int nOrgWidth, float rnOrgWidth, int iGeneSpace, float rnPairs, float gapOpenCost, float gapExtendCost)
{
	// Block index
	int bx = blockIdx.x;
	//    int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int curPos = ty*nWidth + bx*blockDim.x + tx;
#if 0
	int orgIdx = curPos%nPopulationWidth / nOrgWidth;
	if (orgIdx >= MSA_GA_POPULATION_SIZE)  return;
	int seqIdx = curPos / nPopulationWidth;
	int posIdx = curPos - seqIdx*nPopulationWidth - orgIdx*nOrgWidth;
#endif
	int seqIdx, r;
	devMOD(curPos, nPopulationWidth, rnPopulationWidth, seqIdx, r);
	int orgIdx = r*rnOrgWidth;
	int posIdx = curPos - seqIdx*nPopulationWidth - orgIdx*nOrgWidth;

	//if the unit is invalid, we simply set the score of the unit to zero and return.
	if (d_population[curPos] == MSA_GA_CUDA_SPACE)
	{
		d_score[curPos] = 0;
		return;
	}

	//gap panalty,must calculate before the pair score!!!!!
	float panaltyScore = 0;
	//gapOpenCost = gapOpenCost + log10(d_populationLen[orgIdx]);
	if (d_population[curPos] == iGeneSpace)
	{
#if 0
		int nGapPair = 1;
		if (orgIdx < nSeq - 1)
		{
			for (int i = seqIdx + 1; i < nSeq; ++i)
			{
				if (d_population[i*nPopulationWidth + orgIdx*nOrgWidth + posIdx] == iGeneSpace)  ++nGapPair;
			}
		}

		bool bLeftGap=false, bRightGap=false;
		int curPosIdx = posIdx;
		for (int i = 1; i < 8; ++i)
		{
			--curPosIdx;
			if (curPosIdx <= 0) break;
			if (d_population[seqIdx*nPopulationWidth + orgIdx*nOrgWidth + curPosIdx - 1] == iGeneSpace && d_population[seqIdx*nPopulationWidth + orgIdx*nOrgWidth + curPosIdx] != iGeneSpace)
			{
				bLeftGap = true;
				break;
			}
		}
		curPosIdx = posIdx;
		for (int i = 1; i < 8; ++i)
		{
			++curPosIdx;
			if (curPosIdx >= d_populationLen[orgIdx]) break;
			if (d_population[seqIdx*nPopulationWidth + orgIdx*nOrgWidth + curPosIdx + 1] == iGeneSpace && d_population[seqIdx*nPopulationWidth + orgIdx*nOrgWidth + curPosIdx] != iGeneSpace)
			{
				bRightGap = true;
				break;
			}
		}
		if (bLeftGap || bRightGap)  gapExtendCost = gapExtendCost*2.0f;
#endif
		int nGapPair = 1;
		if (posIdx == 0)
		{//the start of the seq is a GAP
			panaltyScore = gapOpenCost;// / nGapPair;
		}
		else
		{
			if (d_population[curPos - 1] == iGeneSpace)
			{//gap extension
				panaltyScore = gapExtendCost;
			}
			else
			{//gap open
				panaltyScore = gapOpenCost;// / nGapPair;
			}
		}
	}
	d_score[curPos] = panaltyScore / nSeq;// *nSeq*(nSeq - 1)*0.5f;

	if (seqIdx == nSeq - 1)
	{//��������һ�����У��������жԵ÷�(pair score)�Ѿ���ǰ�������н����˼��㣬����Ҫ�������һ�����е����жԵ÷֣�
		return;
	}

	//pair score
	float colScore = 0;
	short	aCode = d_population[curPos];
	for (int i = seqIdx + 1; i<nSeq; ++i)
	{
		int idx = i*nPopulationWidth + orgIdx*nOrgWidth + posIdx;
		short	bCode = d_population[idx];
		if (aCode != iGeneSpace && bCode != iGeneSpace)
		{
			if (aCode<bCode)
			{
				colScore += d_subMat[SubMatPos(bCode, aCode)] * rnPairs;// *d_seqWeight[curPos] * d_seqWeight[idx] * rnPairs;//
			}
			else
			{
				colScore += d_subMat[SubMatPos(aCode, bCode)] * rnPairs;// *d_seqWeight[curPos] * d_seqWeight[idx] * rnPairs;//
			}
		}
	}

	d_score[curPos] += colScore;
}

template <int BLOCK_SIZE> __global__ void
MSAGA_SPS_UNIT_CUDA(short *d_score, short *d_population, short *d_subMat, int nHeight, int nWidth, int blockWidth, int GAPCODE, int gapOpenCost, int gapExtendCost )
{
	// Block index
    int bx = blockIdx.x;
//    int by = blockIdx.y;
	
	// Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	int curPos = ty*nWidth+bx*blockWidth+tx;
	
	//if the unit is invalid, we simply set the score of the unit to zero and return.
	if (d_population[curPos] == MSA_GA_CUDA_SPACE)
	{
		d_score[curPos] = 0;
		return;
	}

	//gap panalty,must calculate before the pair score!!!!!
	short panaltyScore = 0;
	if( d_population[curPos]==GAPCODE )
	{
		if(tx==0)
		{//the start of the seq is a GAP
			panaltyScore = gapOpenCost;
		}
		else
		{
			if( d_population[curPos-1]==GAPCODE )
			{//gap extension
				panaltyScore = gapExtendCost;
			}
			else
			{//gap open
				panaltyScore = gapOpenCost;
			}
		}
	}
	d_score[curPos] = panaltyScore;

	 if(ty==nHeight-1)  
	 {//��������һ�����У��������жԵ÷�(pair score)�Ѿ���ǰ�������н����˼��㣬����Ҫ�������һ�����е����жԵ÷֣�
		 return;
	 }
	
	//pair score
	short colScore=0;
	short	aCode = d_population[curPos];
	for( int i=ty+1; i<nHeight; ++i )
	{
		short	bCode =  d_population[i*nWidth+bx*blockWidth+tx];
		if( aCode!=GAPCODE && bCode!=GAPCODE )
		{
			if ( aCode<bCode )
			{
				colScore += d_subMat[SubMatPos(bCode, aCode)];
			}
			else
			{
				colScore += d_subMat[SubMatPos(aCode, bCode)];
			}
		}
	}

	d_score[curPos] += colScore;
}

__global__ void MSAGA_SPS_COLSUM_CUDA_UNLIMIT(float *d_score_colsum, float *d_score, int nSeq, int nWidth, int nPopuationNum, int nPopulationWidth, int nOrgWidth, float rnOrgWidth)
{
	// Block index
	int bx = blockIdx.x;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int curPos = ty*nWidth + bx*blockDim.x + tx;
	int orgIdx = curPos*rnOrgWidth;
	if (orgIdx >= nPopuationNum)  return;
	int posIdx = curPos - orgIdx*nOrgWidth;

	float score_col_sum = 0;

	for (int i = 0; i<nSeq; ++i)
	{
		score_col_sum += d_score[i*nPopulationWidth + orgIdx*nOrgWidth + posIdx];
	}

	d_score_colsum[curPos] = score_col_sum;
}

template <int BLOCK_SIZE> __global__ void
MSAGA_SPS_COLSUM_CUDA(short *d_score_colsum, short *d_score, int nHeight, int nWidth, int blockWidth)
{
	// Block index
    int bx = blockIdx.x;
//    int by = blockIdx.y;
	
	// Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	short score_col_sum = 0;

	for( int i=0; i<nHeight; ++i )
	{
		score_col_sum += d_score[i*nWidth+bx*blockWidth+tx];
	}

	//__syncthreads();

	d_score_colsum[ty*nWidth+bx*blockWidth+tx] = score_col_sum;
}

__global__ void MSAGA_SPS_ORGAN_CUDA_UNLIMIT(float *d_score_organ, float *d_score_colsum, int nWidth, int nOrgWidth, int nPopulationNum)
{
	// Block index
	int bx = blockIdx.x;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int curPos = ty*nWidth + bx*blockDim.x + tx;
	if (curPos >= nPopulationNum)  return;

	float score_organ = 0;
	for (int i = 0; i<nOrgWidth; ++i)
	{
		score_organ += d_score_colsum[curPos*nOrgWidth + i];
	}

	d_score_organ[curPos] = score_organ;
}

template <int BLOCK_SIZE> __global__ void
MSAGA_SPS_ORGAN_CUDA(short *d_score_organ, short *d_score_colsum, int nSeqMaxLen, int blockWidth)
{
	// Block index
    int bx = blockIdx.x;
//    int by = blockIdx.y;
	
	// Thread index
    int tx = threadIdx.x;
//    int ty = threadIdx.y;

	short score_organ = 0;
	for( int i=0; i<nSeqMaxLen; ++i )
	{
		score_organ += d_score_colsum[(bx*blockWidth+tx)*nSeqMaxLen+i];
	}

	//__syncthreads();

	d_score_organ[bx*blockWidth+tx] = score_organ;
}

template <int BLOCK_SIZE> __global__ void
MSAGA_ALIGN_CUDA_UNLIMIT(float *d_score_organ_align, uint *d_organ_index_align, float *d_score_organ, int len, int nWidth)
{
	// Block index
	int bx = blockIdx.x;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int orgIdx = ty*nWidth + bx*blockDim.x + tx;
	if (orgIdx<len)
	{
		d_score_organ_align[orgIdx] = d_score_organ[orgIdx];
		d_organ_index_align[orgIdx] = orgIdx;
	}
	else
	{
		d_score_organ_align[orgIdx] = -1e4;
	}
}

template <int BLOCK_SIZE> __global__ void
MSAGA_ALIGN_CUDA(short *d_score_organ_align, uint *d_organ_index_align, short *d_score_organ, int len)
{
	// Block index
    int bx = blockIdx.x;
//    int by = blockIdx.y;
	
	// Thread index
    int tx = threadIdx.x;
//    int ty = threadIdx.y;

	int curPos = bx*blockDim.x+tx;
	if( curPos<len )
	{
		d_score_organ_align[curPos] = d_score_organ[curPos];
		d_organ_index_align[curPos] = curPos;
	}
	else
	{
		d_score_organ_align[curPos] = -1e4;
	}
}

//the arrangement only occur at the changed organ,
//so the gap recorder only record the changed organ.
template <int BLOCK_SIZE> __global__ void
MSAGA_GAP_RECORDER(bool bRecomb, short *d_gapRecorder, short *d_population, float* d_rand_organ_idx, float fChance, int GAPCODE, int nWidth, int nSeq, int nPopulationNum)
{
	// Block index
    int bx = blockIdx.x;
//    int by = blockIdx.y;
	
	// Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	if ( bx==0 )
	{
		return;
	}

	int  curIdx = -1;
	if (bRecomb)
	{
		//r=(MSA_GA_POPULATION_SIZE - 1)%2
		int q, r;
		devMOD(nPopulationNum - 1, 2, 0.5f, q, r);
		if (r = 0 && (bx == nPopulationNum - 1))
		{//�ܸ���Ϊ���������һ�����岻����
			return;
		}
		else
		{
			int  curOrder = bx;
			//r=bx%2
			int q, r;
			devMOD(bx, 2, 0.5f, q, r);
			if (r == 0)//�����ǰ������ż����Ÿ���
			{
				curOrder = bx - 1;//ʹ����������������Ž��м���
			}
			curIdx = curOrder / 2;//half of the population size
		}
	}
	else
	{
		curIdx = bx;
	}

	if (d_rand_organ_idx[curIdx]<fChance)//ֻ����䶯�˵ĸ���
	{
		bool   bGapColumn = true;
		for( int i=0; i<nSeq; ++i )
		{//�Ե�ǰ�е����вл����б���������Ƿ�ȫ�ж�ΪGAP
			if(d_population[bx*blockDim.x+i*nWidth+tx] != GAPCODE&&d_population[bx*blockDim.x+i*nWidth+tx] != MSA_GA_CUDA_SPACE)
			{
				bGapColumn = false;
			}
		}
		if( bGapColumn )
		{//�����ǰ��ȫ�ж�ΪGAP����δ�ÿռ䣬���Ǵ���Ϊ0
			d_gapRecorder[bx*blockDim.x+tx] = 0;
		}
		else
		{//�����ǰ�в�ȫΪGAP�����Ǵ���Ϊ1
			d_gapRecorder[bx*blockDim.x+tx] = 1;
		}
	}
}

template <int BLOCK_SIZE> __global__ void
MSAGA_GAP_RANGE( short *d_gapRange, short *d_GapRecorder, float* d_rand_organ_idx, float fChance  )
{
	// Block index
    int bx = blockIdx.x;
//    int by = blockIdx.y;
	
	// Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	if( d_rand_organ_idx[bx]<fChance )
	{
		int countGap=0;
		for( int i=0; i<tx; ++i )
		{
			int prePos=bx*blockDim.x + i;
			int curPos=prePos+1;
			if( d_GapRecorder[prePos]==1 && d_GapRecorder[curPos]==0 )
			{
				++countGap;
			}
		}
		d_gapRange[bx*blockDim.x+tx] = countGap;
	}

}

template <int BLOCK_SIZE> __global__ void
MSAGA_GAP_LEN( short *d_GapLenRecorder, short *d_GapRecorder, float* d_rand_organ_idx, float fChance, int nWidth )
{
	// Block index
    int bx = blockIdx.x;
//    int by = blockIdx.y;
	
	// Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	if( d_rand_organ_idx[bx]<fChance  )
	{
		int gapNum=tx+1;
		int count=0;
		int prePos=0;
		int curPos=prePos+1;
		int gapStartPos=0;
		int gapLen=0;
		while( count<gapNum && curPos<nWidth && d_GapRecorder[curPos] !=MSA_GA_CUDA_SPACE )
		{
			if( d_GapRecorder[prePos]==1 && d_GapRecorder[curPos]==0 )
			{
				gapStartPos = curPos;
			}
			if( d_GapRecorder[prePos]==0 && d_GapRecorder[curPos]==1 )
			{
				gapLen += curPos-gapStartPos;
				++count;
			}
			++prePos;
			++curPos;
		}
		d_GapLenRecorder[bx*blockDim.x+tx]=gapLen;
	}
}

template <int BLOCK_SIZE> __global__ void
MSAGA_CLEAR_POPULATON_NEXT_GEN( short *d_populationNextGen, float* d_rand_organ_idx, float fChance, int nWidth )
{
	// Block index
    int bx = blockIdx.x;
//    int by = blockIdx.y;
	
	// Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	if(d_rand_organ_idx[bx]<fChance)//����䶯�˵ĸ���
	{
		d_populationNextGen[bx*blockDim.x+ty*nWidth+tx]=MSA_GA_CUDA_SPACE;
	}
}

__global__	void	MSAGA_DELETE_SPACE_AT_TAIL_UNLIMIT(bool bRecomb, short *d_populationNextGen, short *d_population, float* d_rand_organ_idx, float fChance, int nWidth, int nPopulationWidth, float rnPopulationWidth, int nOrgWidth, float rnOrgWidth, int nSeq, int iGeneSpace, int nCellNum, int nPopulationNum)
{
	// Block index
	int bx = blockIdx.x;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int curPos = bx*blockDim.x + ty*nWidth + tx;
	if(curPos>=nCellNum)  return;
#if 0
	int orgIdx = int(curPos%nPopulationWidth / nOrgWidth);
	if (orgIdx >= MSA_GA_POPULATION_SIZE)   return;
	int seqIdx = int(curPos / nPopulationWidth);
	int posIdx = curPos - seqIdx*nPopulationWidth - orgIdx*nOrgWidth;
#endif
	int seqIdx = floor(curPos*rnPopulationWidth);
	int i = curPos - seqIdx*nPopulationWidth;
	int orgIdx = i*rnOrgWidth;
	int posIdx = curPos - seqIdx*nPopulationWidth - orgIdx*nOrgWidth;

	if (orgIdx == 0)
	{
		d_populationNextGen[curPos] = d_population[curPos];
		return;
	}

	int  curIdx = -1;
	if (bRecomb)
	{
		int q, r;
		devMOD(nPopulationNum - 1, 2, 0.5f, q, r);
		if (r!= 0 && orgIdx == (nPopulationNum - 1))
		{//�ܸ���Ϊ���������һ������ֱ�Ӹ���
			d_populationNextGen[curPos] = d_population[curPos];
			return;
		}
		else
		{
			int  curOrder = orgIdx;
			int q, r;
			devMOD(orgIdx, 2, 0.5f, q, r);
			if (r == 0)//�����ǰ������ż����Ÿ���
			{
				curOrder = orgIdx - 1;//ʹ����������������Ž��м���
			}
			curIdx = curOrder * 0.5f;//curIdx = curOrder / 2;//half of the population size
		}
	}
	else
	{
		curIdx = orgIdx;
	}

	if (d_rand_organ_idx[curIdx]<fChance)//����䶯�˵ĸ���
	{
		for (int i = 0; i < nSeq; i++)
		{
			if( d_population[orgIdx*nOrgWidth + i*nPopulationWidth + posIdx] != iGeneSpace)
			{//��������вл�����MSA_GA_CUDA_SPACE����ֱ�Ӹ���
				d_populationNextGen[curPos] = d_population[curPos];
				return;
			}
		}
		d_populationNextGen[curPos] = MSA_GA_CUDA_SPACE;
	}
	else//û�б䶯�ĸ���ֱ�Ӹ���
	{
		d_populationNextGen[curPos] = d_population[curPos];
	}
}

__global__	void	MSAGA_DELETE_SPACE_UNLIMIT( bool bRecomb, short *d_populationNextGen, short *d_population, float* d_rand_organ_idx, float fChance, short *d_gapRecorder, int nWidth, int nPopulationNum, int nPopulationWidth, float rnPopulationWidth, int nOrgWidth, float rnOrgWidth, int nCellNum )
{
	// Block index
	int bx = blockIdx.x;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int curPos = bx*blockDim.x+ty*nWidth+tx;
	if (curPos >= nCellNum)  return;
#if 0
	int orgIdx = int(curPos%nPopulationWidth/nOrgWidth);
	if (orgIdx >= MSA_GA_POPULATION_SIZE)   return;
	int seqIdx = int(curPos/nPopulationWidth);
	int posIdx = curPos-seqIdx*nPopulationWidth-orgIdx*nOrgWidth;
#endif
	int seqIdx = floor(curPos*rnPopulationWidth);
	int i = curPos - seqIdx*nPopulationWidth;
	int orgIdx = i*rnOrgWidth;
	int posIdx = curPos - seqIdx*nPopulationWidth - orgIdx*nOrgWidth;

	if ( orgIdx==0 )
	{
		d_populationNextGen[curPos]=d_population[curPos];
		return;
	}

	int  curIdx = -1;
	if (bRecomb)
	{
		int q, r;
		devMOD(nPopulationNum - 1, 2, 0.5f, q, r);
		if (r != 0 && orgIdx == (nPopulationNum - 1))
		{//�ܸ���Ϊ���������һ������ֱ�Ӹ���
			d_populationNextGen[curPos] = d_population[curPos];
			return;
		}
		else
		{
			int  curOrder = orgIdx;
			devMOD(orgIdx, 2, 0.5f, q, r);
			if (r == 0)//�����ǰ������ż����Ÿ���
			{
				curOrder = orgIdx - 1;//ʹ����������������Ž��м���
			}
			curIdx = curOrder / 2;//half of the population size
		}
	}
	else
	{
		curIdx = orgIdx;
	}

	if (d_rand_organ_idx[curIdx]<fChance)//����䶯�˵ĸ���
	{
		if (d_gapRecorder[orgIdx*nOrgWidth + posIdx] != 0)
		{//��ǰ�����вл�����
			int gapLen=0;
			for( int i=0; i<posIdx; ++i )
			{
				if( d_gapRecorder[orgIdx*nOrgWidth+i]==0 )
				{
					++gapLen;
				}
			}
			//����ǰλ�õ������ƶ����ո�֮ǰ	
			d_populationNextGen[curPos-gapLen]=d_population[curPos];
		}
	}
	else//û�б䶯�ĸ���ֱ�Ӹ���
	{
		d_populationNextGen[curPos]=d_population[curPos];
	}
}

template <int BLOCK_SIZE> __global__ void
MSAGA_DELETE_SPACE(  short *d_populationNextGen, short *d_population, float* d_rand_organ_idx, float fChance, short *d_gapRecorder, int nWidth )
{
	// Block index
    int bx = blockIdx.x;
//    int by = blockIdx.y;
	
	// Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	int pos=bx*blockDim.x+ty*nWidth+tx;
	if(d_rand_organ_idx[bx]<fChance)//����䶯�˵ĸ���
	{
		if( d_gapRecorder[bx*blockDim.x+tx]==1 )//��ǰ�в��ǿո��Ҳ���δ�ÿռ�
		{
			int gapLen=0;
			for( int i=0; i<tx; ++i )
			{
				if( d_gapRecorder[bx*blockDim.x+i]==0 )
				{
					++gapLen;
				}
			}
			//����ǰλ�õ������ƶ����ո�֮ǰ	
			d_populationNextGen[pos-gapLen]=d_population[pos];
		}
	}
	else//û�б䶯�ĸ���ֱ�Ӹ���
	{
		d_populationNextGen[pos]=d_population[pos];
	}
}

__global__	void		MSAGA_FILL_SPACE_AT_TAIL_UNLIMIT( bool bRecomb, short *d_populationNextGen, short *d_population, float* d_rand_organ_idx, float fChance, int nWidth, short SPACE,  int nOrgWidth, float rnOrgWidth, int nPopulationWidth, float rnPopulationWidth, int nSeq, int nCellNum, int nPopulationNum )
{
	// Block index
	int bx = blockIdx.x;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int curPos = bx*blockDim.x+ty*nWidth+tx;
	if(curPos>=nCellNum)  return;
#if 0
	int orgIdx = int(curPos%nPopulationWidth/nOrgWidth);
	if (orgIdx >= MSA_GA_POPULATION_SIZE)  return;
	int seqIdx = int(curPos/nPopulationWidth);
	int posIdx = curPos-seqIdx*nPopulationWidth-orgIdx*nOrgWidth;
#endif
	int seqIdx = floor(curPos*rnPopulationWidth);
	int i = curPos - seqIdx*nPopulationWidth;
	int orgIdx = i*rnOrgWidth;
	int posIdx = curPos - seqIdx*nPopulationWidth - orgIdx*nOrgWidth;


	if ( orgIdx==0 )
	{
		d_populationNextGen[curPos] = d_population[curPos];
		return;
	}

	short curIdx = -1;
	if (bRecomb)
	{
		int d, r;
		devMOD(nPopulationNum - 1, 2, 0.5f, d, r);
		if (r != 0 && orgIdx == (nPopulationNum - 1))
		{//�ܸ���Ϊ���������һ������ֱ�Ӹ���
			d_populationNextGen[curPos] = d_population[curPos];
			return;
		}
		else
		{
			int  curOrder = orgIdx;
			devMOD(orgIdx, 2, 0.5f, d, r);
			if (r == 0)//�����ǰ������ż����Ÿ���
			{
				curOrder = orgIdx - 1;//ʹ����������������Ž��м���
			}
			//curIdx = curOrder / 2;//half of the population size
			curIdx = curOrder *0.5f;
		}
	}
	else
	{
		curIdx = orgIdx;
	}

	if (d_rand_organ_idx[curIdx]<fChance)//����䶯�˵ĸ���
	{
		if( d_population[curPos]==MSA_GA_CUDA_SPACE )
		{
			bool  bAddSpace = false;
			for( int i=0; i<nSeq; ++i )
			{
				if( d_population[orgIdx*nOrgWidth+i*nPopulationWidth+posIdx]!=MSA_GA_CUDA_SPACE )
				{
					bAddSpace = true;
					break;
				}
			}

			if(bAddSpace)
			{
				d_populationNextGen[curPos] = SPACE;
			}
			else
			{
				d_populationNextGen[curPos] = MSA_GA_CUDA_SPACE;
			}
		}
		else
		{
			d_populationNextGen[curPos] = d_population[curPos];
		}
	}
	else
	{
		d_populationNextGen[curPos] = d_population[curPos];
	}
}

template <int BLOCK_SIZE> __global__ void
MSAGA_FILL_SPACE_AT_TAIL( short *d_populationNextGen, short *d_population, float* d_rand_organ_idx, float fChance, int nWidth, short SPACE )
{
	// Block index
    int bx = blockIdx.x;
//    int by = blockIdx.y;
	
	// Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	int curPos = bx*blockDim.x+ty*nWidth+tx;
	if(d_rand_organ_idx[bx]<fChance)//����䶯�˵ĸ���
	{
		if( d_population[curPos]==MSA_GA_CUDA_SPACE )
		{
			bool  bAddSpace = false;
			for( int i=0; i<blockDim.y; ++i )
			{
				if( d_population[bx*blockDim.x+i*nWidth+tx]!=MSA_GA_CUDA_SPACE )
				{
					bAddSpace = true;
					break;
				}
			}

			if(bAddSpace)
			{
				d_populationNextGen[curPos] = SPACE;
			}
			else
			{
				d_populationNextGen[curPos] = d_population[curPos];
			}
		}
		else
		{
			d_populationNextGen[curPos] = d_population[curPos];
		}
	}
	else
	{
		d_populationNextGen[curPos] = d_population[curPos];
	}
}

template <int BLOCK_SIZE> __global__ void
MSAGA_DELETE_SPACE_COLUMN( short *d_populationNextGen, short *d_population, short *d_gapRecorder, float *d_rand_organ_idx, float fChance, int nWidth )
{
	// Block index
    int bx = blockIdx.x;
//    int by = blockIdx.y;
	
	// Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	int curPos = bx*blockDim.x+ty*nWidth+tx;
	if(d_rand_organ_idx[bx]<fChance)//����䶯�˵ĸ���
	{
		int nGap=0;
		for( int i=0; i<=tx; ++i )
		{//���[0��tx]������ȫ�ǿ�λ����ĿnGap��Tx����Tx+nGapλ�õ��������.
			if( d_gapRecorder[bx*blockDim.x+i]==0 )  ++nGap; 
		}
		if( tx+nGap<blockDim.x )
		{
			d_populationNextGen[curPos] = d_population[curPos+nGap];
		}
		else
		{//���tx+nGap��ָλ�ó������������г��ȣ�����δʹ�ÿռ�ռλ�����
			d_populationNextGen[curPos] = MSA_GA_CUDA_SPACE;
		}
	}
	else
	{
		d_populationNextGen[curPos] = d_population[curPos];
	}
}

template <int BLOCK_SIZE> __global__ void
MSAGA_LOCATE_GAP( short* d_Gap_Tx, short*  d_mark, float* d_rand_Tx_number, float *d_rand_organ_idx,  int seqMaxLen, int nWidth, float fChance, int GAPCODE )
{
	// Block index
//    int bx = blockIdx.x;
//    int by = blockIdx.y;
	
	// Thread index
    int tx = threadIdx.x;
//    int ty = threadIdx.y;

	//only handle the randomly chosen organs.
	int n=0;
	d_Gap_Tx[tx]=-1;
	if(d_rand_organ_idx[tx]<fChance)
	{
		for( int i=0; i<seqMaxLen; ++i )
		{
			if( d_mark[tx*seqMaxLen+i]==1 )
			{
				++n;
			}
		}
		int idx=d_rand_Tx_number[tx]*n;//�����ܵ�������λ���������ѡ��һ��������λ��Ϊ��������
		n=-1;
		for( int i=0; i<seqMaxLen; ++i )
		{
			if( d_mark[tx*seqMaxLen+i]==1 )
			{
				++n;
			}
			if( n==idx )//�ҵ�Ӧ�ò���GAP��������λ��λ��
			{
				d_Gap_Tx[tx] = i;
				break;
			}
		}
		
	}
}

template <int BLOCK_SIZE> __global__ void
MSAGA_LOCATE_GAP_UNLIMIT(short* d_Gap_Tx, short*  d_mark, float* d_rand_Tx_number, float *d_rand_organ_idx, int seqMaxLen, int nWidth, float fChance, int GAPCODE, int nCellNum)
{
	// Block index
	    int bx = blockIdx.x;
	//    int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int orgIdx = bx*blockDim.x + ty*nWidth + tx;
	if (orgIdx >= nCellNum)  return;

	//only handle the randomly chosen organs.
	int n = 0;
	d_Gap_Tx[orgIdx] = -1;
	if (d_rand_organ_idx[orgIdx]<fChance)
	{
		for (int i = 0; i<seqMaxLen; ++i)
		{
			if (d_mark[orgIdx*seqMaxLen + i] == 1)
			{
				++n;
			}
		}
		int idx = d_rand_Tx_number[orgIdx] * n;//�����ܵ�������λ���������ѡ��һ��������λ��Ϊ��������
		n = -1;
		for (int i = 0; i<seqMaxLen; ++i)
		{
			if (d_mark[orgIdx*seqMaxLen + i] == 1)
			{
				++n;
			}
			if (n == idx)//�ҵ�Ӧ�ò���GAP��������λ��λ��
			{
				d_Gap_Tx[orgIdx] = i;
				break;
			}
		}

	}
}

__global__	void	MSAGA_MARK_GAP_START_UNLIMIT( short*  d_mark, short* d_population, float *d_rand_organ_idx,  float* d_rand_Ty, int nSeqNum, int nWidth, float fChance, int GAPCODE,  int nOrgWidth, float rnOrgWidth, int nPopulationWidth, float rnPopulationWidth, int nCellNum )
{
	// Block index
    int bx = blockIdx.x;
	
	// Thread index
    int tx = threadIdx.x;
	int ty = threadIdx.y;

	int curPos = bx*blockDim.x+ty*nWidth+tx;
	if( curPos>=nCellNum )  return;
#if 0
	int orgIdx = curPos%nPopulationWidth/nOrgWidth;
	if (orgIdx >= MSA_GA_POPULATION_SIZE)  return;
	int seqIdx = curPos/nPopulationWidth;
	int posIdx = curPos-seqIdx*nPopulationWidth-orgIdx*nOrgWidth;
#endif
	int seqIdx = floor(curPos*rnPopulationWidth);
	int i = curPos - seqIdx*nPopulationWidth;
	int orgIdx = i*rnOrgWidth;
	int posIdx = curPos - seqIdx*nPopulationWidth - orgIdx*nOrgWidth;

	//only handle the randomly chosen organs.
	if(d_rand_organ_idx[orgIdx]<fChance)
	{
		int idxTy = d_rand_Ty[orgIdx]*(nSeqNum-1);//ȷ���ø������ĸ����н��б��
		int curOrganPos=orgIdx*nOrgWidth+idxTy*nPopulationWidth+posIdx;
		if(posIdx==0)//��ǰλ��������ͷ��
		{
			if( d_population[curOrganPos]==GAPCODE )
			{
				d_mark[orgIdx*nOrgWidth + posIdx] = 1;
			}
			else
			{
				d_mark[orgIdx*nOrgWidth + posIdx] = 0;
			}
		}
		else//��ǰλ�ò�������ͷ��ʱ
		{
			if(  d_population[curOrganPos]==GAPCODE && d_population[curOrganPos-1]!=GAPCODE )//��ǰλ��ΪGAP����ǰһλ�ò���GAP����ȷ����ǰλ��Ϊ��λ��ʼλ��
			{
				d_mark[orgIdx*nOrgWidth + posIdx] = 1;
			}
			else
			{
				d_mark[orgIdx*nOrgWidth + posIdx] = 0;
			}
		}
	}
}

template <int BLOCK_SIZE> __global__ void
MSAGA_MARK_GAP_START( short*  d_mark, short* d_population, float *d_rand_organ_idx,  float* d_rand_Ty, int nSeqNum, int nWidth, float fChance, int GAPCODE )
{
	// Block index
    int bx = blockIdx.x;
//    int by = blockIdx.y;
	
	// Thread index
    int tx = threadIdx.x;
//    int ty = threadIdx.y;

	//only handle the randomly chosen organs.
	if(d_rand_organ_idx[bx]<fChance)
	{
		int curPos = bx*blockDim.x+tx;
		int idxTy = d_rand_Ty[bx]*(nSeqNum-1);//ȷ���ø������ĸ����н��б��
		int curOrganPos=bx*blockDim.x+idxTy*nWidth+tx;
		if(tx==0)//��ǰλ��������ͷ��
		{
			if( d_population[curOrganPos]==GAPCODE )
			{
				d_mark[curPos] = 1;
			}
			else
			{
				d_mark[curPos] = 0;
			}
		}
		else//��ǰλ�ò�������ͷ��ʱ
		{
			if(  d_population[curOrganPos]==GAPCODE && d_population[curOrganPos-1]!=GAPCODE )//��ǰλ��ΪGAP����ǰһλ�ò���GAP����ȷ����ǰλ��Ϊ��λ��ʼλ��
			{
				d_mark[curPos] = 1;
			}
			else
			{
				d_mark[curPos] = 0;
			}
		}
	}
}

__global__ void  MSAGA_RECOMB_VERTICAL_UNLIMIT(  short *d_populationNextGen, short *d_population, short* d_pOrgIdx0, short* d_pOrgIdx1, short* d_pRecombPos0, short* d_pRecombPos1, float *d_rand_organ_idx, float  fRecombVertical, short *d_Pos1_min, short *d_Pos1_max, int geneSpace, int nOrgWidth, float rnOrgWidth, int nPopulationWidth, float rnPopulationWidth, int nPopulation, int nWidth, int nCellNum  )
{
	// Block index    
	int bx = blockIdx.x;
	//    int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int curPos = ty*nWidth+bx*blockDim.x+tx;
	if(curPos>=nCellNum)  return;
	int seqIdx = floor(curPos*rnPopulationWidth);
	int i = curPos - seqIdx*nPopulationWidth;
	int orgIdx = i*rnOrgWidth;
	int posIdx = curPos - seqIdx*nPopulationWidth - orgIdx*nOrgWidth;
#if 0
	int orgIdx = curPos%nPopulationWidth/nOrgWidth;
	if( orgIdx>=nPopulation )  return;
	int seqIdx = curPos/nPopulationWidth;
	int posIdx = curPos-seqIdx*nPopulationWidth-orgIdx*nOrgWidth;
#endif

	if(orgIdx==0) 
	{//���Ÿ���ֱ�ӽ�����һ��
		d_populationNextGen[curPos] = d_population[curPos];
		return;
	}	

	int r, d;
	devMOD(nPopulation-1, 2, 0.5f, d, r);
	if (r != 0 && orgIdx == nPopulation - 1)//����������������Ϊ�����������һ������ֱ�ӽ�����һ��
	{
		d_populationNextGen[curPos] = d_population[curPos];
		return;
	}

	int  curOrder=orgIdx;
	devMOD(orgIdx, 2, 0.5f, d, r);
	if( r==0 )//�����ǰ������ż����Ÿ���
	{
		curOrder=orgIdx-1;//ʹ����������������Ž��м���
	}

	int curIdx = curOrder/2;//half of the population size=nRecomb
	//d_pBx[curIdx]=0;
	if(d_rand_organ_idx[curIdx]<fRecombVertical)
	{
		//d_pBx[curIdx]=1;
		int nRecomb = (nPopulation-1)/2;
		if( orgIdx%2==0 )//�����ǰ������ż����Ÿ���
		{
			//d_populationNextGen[curPos] = d_population[curPos];
			if( posIdx<=d_pRecombPos1[curIdx+seqIdx*nRecomb] )
			{//copy directly from organ1,ʹ��P1����Ӧλ������
				d_populationNextGen[curPos] = d_population[d_pOrgIdx1[curIdx]*nOrgWidth+seqIdx*nPopulationWidth+posIdx];
			}
			else if( posIdx>d_Pos1_max[curIdx] )
			{
				int posIdx1=d_pRecombPos0[curIdx]+posIdx-d_Pos1_max[curIdx];
				int copyPos = d_pOrgIdx0[curIdx]*nOrgWidth+seqIdx*nPopulationWidth+posIdx1;
				if( posIdx1<nOrgWidth )
				{
					d_populationNextGen[curPos] = d_population[copyPos];
				}
				else
				{//���P0�������ݾ��Ѿ����ƣ������Ŀռ�ȫ��Ӧ�ñ��Ϊδʹ��
					d_populationNextGen[curPos] = MSA_GA_CUDA_SPACE;
				}
			}
			else
			{//insert gap,����SPACE1���ո�
				d_populationNextGen[curPos] = geneSpace;
			}
		}
		else//�����ǰ������������Ÿ���
		{
			//d_populationNextGen[curPos] = d_population[curPos];
			if( posIdx<=d_pRecombPos0[curIdx] )
			{//copy directly from organ0,ֱ�Ӹ��Ƹ���P0��Ӧλ�õ�����
				d_populationNextGen[curPos] = d_population[d_pOrgIdx0[curIdx]*nOrgWidth+seqIdx*nPopulationWidth+posIdx];
			}
			else if( posIdx>(d_pRecombPos0[curIdx]+d_pRecombPos1[curIdx+seqIdx*nRecomb]-d_Pos1_min[curIdx]) )
			{
				int posIdx1=d_pRecombPos1[curIdx+seqIdx*nRecomb]+posIdx-(d_pRecombPos0[curIdx]+d_pRecombPos1[curIdx+seqIdx*nRecomb]-d_Pos1_min[curIdx]);
				int copyPos = d_pOrgIdx1[curIdx]*nOrgWidth+seqIdx*nPopulationWidth+posIdx1;
				if( posIdx1<nOrgWidth )
				{	
					d_populationNextGen[curPos] =d_population[copyPos];
				}
				else
				{
					d_populationNextGen[curPos] = MSA_GA_CUDA_SPACE;
				}
			}
			else
			{//insert gap������recombPos1-min(recombPos1)���ո�
				d_populationNextGen[curPos] = geneSpace;
			}
		}
	}
	else//�����㴹ֱ�ӽ������ĸ���ֱ�Ӹ���
	{
		d_populationNextGen[curPos] = d_population[curPos];
	}
}

template <int BLOCK_SIZE> __global__ void
MSAGA_RECOMB_VERTICAL(  short *d_populationNextGen, short *d_population, short* d_pOrgIdx0, short* d_pOrgIdx1, short* d_pRecombPos0, short* d_pRecombPos1, float *d_rand_organ_idx, float  fRecombVertical, int nWidth,  int nPopulation, int organNum, short *d_Pos1_min, short *d_Pos1_max, int GENESPACE, short NONE )
{
	// Block index    
    int bx = blockIdx.x;
//    int by = blockIdx.y;
	
	// Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	int curPos = ty*nWidth+bx*blockDim.x+tx;
	if(bx==0) 
	{//���Ÿ���ֱ�ӽ�����һ��
		d_populationNextGen[curPos] = d_population[curPos];
		return;
	}	

	//!!!!!!!!!!!!!!!!!!!!!!!!!��Ҫ������ż���壬�ֱ�����������//	
	if( (nPopulation-1)%2!=0 && bx==nPopulation-1 )//����������������Ϊ�����������һ������ֱ�ӽ�����һ��
	{
		d_populationNextGen[curPos] = d_population[curPos];
		return;
	}

	int  curOrder=bx;
	if( bx%2==0 )//�����ǰ������ż����Ÿ���
	{
		curOrder=bx-1;//ʹ����������������Ž��м���
	}

	int curIdx = curOrder/2;//half of the population size=nRecomb
	//d_pBx[curIdx]=0;
	if(d_rand_organ_idx[curIdx]<fRecombVertical)
	{
		//d_pBx[curIdx]=1;
		int nRecomb = (nPopulation-1)/2;
		if( bx%2==0 )//�����ǰ������ż����Ÿ���
		{
			//d_populationNextGen[curPos] = d_population[curPos];
			if( tx<=d_pRecombPos1[curIdx+ty*nRecomb] )
			{//copy directly from organ1,ʹ��P1����Ӧλ������
				d_populationNextGen[curPos] = d_population[d_pOrgIdx1[curIdx]*blockDim.x+ty*nWidth+tx];
			}
			else if( tx>d_Pos1_max[curIdx] )
			{
				int tx1=d_pRecombPos0[curIdx]+tx-d_Pos1_max[curIdx];
				int copyPos = d_pOrgIdx0[curIdx]*blockDim.x+ty*nWidth+tx1;
				if( tx1<blockDim.x )
				{
						d_populationNextGen[curPos] = d_population[copyPos];
				}
				else
				{//���P0�������ݾ��Ѿ����ƣ������Ŀռ�ȫ��Ӧ�ñ��Ϊδʹ��
						d_populationNextGen[curPos] = GENESPACE;
				}
			}
			else
			{//insert gap,����SPACE1���ո�
				d_populationNextGen[curPos] = GENESPACE;
			}
		}
		else//�����ǰ������������Ÿ���
		{
			//d_populationNextGen[curPos] = d_population[curPos];
			if( tx<=d_pRecombPos0[curIdx] )
			{//copy directly from organ0,ֱ�Ӹ��Ƹ���P0��Ӧλ�õ�����
				d_populationNextGen[curPos] = d_population[d_pOrgIdx0[curIdx]*blockDim.x+ty*nWidth+tx];
			}
			else if( tx>(d_pRecombPos0[curIdx]+d_pRecombPos1[curIdx+ty*nRecomb]-d_Pos1_min[curIdx]) )
			{
				int tx1=d_pRecombPos1[curIdx+ty*nRecomb]+tx-(d_pRecombPos0[curIdx]+d_pRecombPos1[curIdx+ty*nRecomb]-d_Pos1_min[curIdx]);
				int copyPos = d_pOrgIdx1[curIdx]*blockDim.x+ty*nWidth+tx1;
				if( tx1<blockDim.x )
				{	
					d_populationNextGen[curPos] =d_population[copyPos];
				}
				else
				{
					d_populationNextGen[curPos] = GENESPACE;
				}
			}
			else
			{//insert gap������recombPos1-min(recombPos1)���ո�
				d_populationNextGen[curPos] = GENESPACE;
			}
		}
	}
	else//�����㴹ֱ�ӽ������ĸ���ֱ�Ӹ���
	{
		d_populationNextGen[curPos] = d_population[curPos];
	}
}

template <int BLOCK_SIZE> __global__ void
MSAGA_RECOMB_VERTICAL_SPACENUM( short *d_SpaceOrg0, short *d_SpaceOrg1, short *d_pOffset_max, short *d_pOffset_min, short *d_pOffset, int nSeqNum, int nWidth )
{
	// Block index
//    int bx = blockIdx.x;
//    int by = blockIdx.y;
	
	// Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	d_SpaceOrg0[ty*nWidth+tx] = abs(d_pOffset_min[tx])+d_pOffset[ty*nWidth+tx];
	d_SpaceOrg1[ty*nWidth+tx] = d_pOffset_max[tx]-d_pOffset[ty*nWidth+tx];
}

template <int BLOCK_SIZE> __global__ void
MSAGA_RECOMB_VERTICAL_POS1_MIN_MAX(short *d_pPos1_min, short *d_pPos1_max, short *d_pRecombPos1, int nSeqNum, float fRecombVertical, float *d_rand_organ_idx, int nWidth)
{
	// Block index
    int bx = blockIdx.x;
//    int by = blockIdx.y;
	
	// Thread index
    int tx = threadIdx.x;
//    int ty = threadIdx.y;

	d_pPos1_min[tx] = -10000;
	d_pPos1_max[tx] = 10000;

	if(d_rand_organ_idx[tx]<fRecombVertical )
	{
		short max = -10000;
		short min = 10000;

		for( int i=0; i<nSeqNum; ++i )
		{
			if( d_pRecombPos1[i*nWidth+tx]>max ) 
			{
				max =  d_pRecombPos1[i*nWidth+tx];
			}
			if( d_pRecombPos1[i*nWidth+tx]<min )
			{
				min =  d_pRecombPos1[i*nWidth+tx];
			}
		}

		d_pPos1_min[tx] = min;
		d_pPos1_max[tx] = max;
	}
}

template <int BLOCK_SIZE> __global__ void
MSAGA_RECOMB_VERTICAL_POS1_MIN_MAX_UNLIMIT(short *d_pPos1_min, short *d_pPos1_max, short *d_pRecombPos1, int nSeqNum, float fRecombVertical, float *d_rand_organ_idx, int nWidth, int nRecomb)
{
	// Block index
	int bx = blockIdx.x;
	//    int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int curPos = bx*blockDim.x + ty*nWidth + tx;
	if (curPos >= nRecomb)   return;

	d_pPos1_min[curPos] = -10000;
	d_pPos1_max[curPos] = 10000;

	if (d_rand_organ_idx[curPos]<fRecombVertical)
	{
		short max = -10000;
		short min = 10000;

		for (int i = 0; i<nSeqNum; ++i)
		{
			if (d_pRecombPos1[i*nRecomb + curPos]>max)
			{
				max = d_pRecombPos1[i*nRecomb + curPos];
			}
			if (d_pRecombPos1[i*nRecomb + curPos]<min)
			{
				min = d_pRecombPos1[i*nRecomb + curPos];
			}
		}

		d_pPos1_min[curPos] = min;
		d_pPos1_max[curPos] = max;
	}
}


__global__ void	MSAGA_RECOMB_VERTICAL_RECOMB_POS1_UNLIMIT(short* d_pRecombPos0, short* d_pRecombPos1, short* d_pOrgIdx0, short* d_pOrgIdx1, float *d_rand_organ_idx, float fRecombVertical, int nWidth, short* d_population, int nPopulationWidth, int nOrgWidth, int nRecomb, float rnRecomb, short iGeneSpace, int nSeq)
{
	//block index
	int bx = blockIdx.x;
	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int curPos = bx*blockDim.x + ty*nWidth+tx;
	if (curPos >= nRecomb*nSeq)  return;

	int curSeq = curPos*rnRecomb;//��ǰλ������Ӧ�ĸ����еĵ�ǰ����
	int curOrg = curPos-curSeq*nRecomb;//��ǰλ������Ӧ�ĸ�����
	d_pRecombPos1[curPos] = -2;
	int n = 0;
	if (d_rand_organ_idx[curOrg] < fRecombVertical)
	{
		//ͳ�Ƹ���ÿ�����дӿ�ͷ���ӽ���֮ǰ�Ĳл�����
		for (int i = 0; i <= d_pRecombPos0[curOrg]; i++)
		{
			if (d_population[d_pOrgIdx0[curOrg] * nOrgWidth + curSeq*nPopulationWidth + i] != iGeneSpace)
			{
				++n;
			}
		}

		//���ҵڶ���������Ӧ�ӽ���λ��
		short j = 0;
		short pos1 = 0;
		if (d_population[d_pOrgIdx0[curOrg] * nOrgWidth + curSeq*nPopulationWidth + 0] == iGeneSpace && n == 0)
		{//�ӵ�һ�����嵽�ӽ��㴦û�вл���ȫΪ�ո���posOrg1Ϊ�ڶ��������е�һ���л�ǰһλ��
			while (d_population[d_pOrgIdx1[curOrg] * nOrgWidth + curSeq*nPopulationWidth + pos1] == iGeneSpace)
			{
				++pos1;
			}
		}
		else
		{
			while (j < n)
			{
				if (d_population[d_pOrgIdx1[curOrg] * nOrgWidth + curSeq*nPopulationWidth + pos1] != iGeneSpace)
				{
					++j;
				}
				++pos1;
			}
		}
		d_pRecombPos1[curPos] = pos1 - 1;
	}
}

__global__ void	MSAGA_RECOMB_VERTICAL_RECOMB_POS0_UNLIMIT(short* d_pRecombPos0, short* d_pOrgIdx0, short* d_pOrgIdx1, float *d_rand_organ_idx, float* d_rand_Tx, float* d_rand_org0, float* d_rand_org1, float fRecombVertical, unsigned int *d_organLen, int nWidth, int nRecomb, int nPopulationNum)
{
	//Block index
	int bx = blockIdx.x;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int curPos = bx*blockDim.x + ty*nWidth + tx;
	if (curPos >= nRecomb)	return;

	d_pRecombPos0[curPos] = -1;
	if (d_rand_organ_idx[curPos] < fRecombVertical)//ÿ��curPos��Ӧһ����ż��ŵĸ���
	{
		//�������һ�Խ��д�ֱ�ӽ��ĸ���
		d_pOrgIdx0[curPos] = short(d_rand_org0[curPos] * (nPopulationNum - 1));
		d_pOrgIdx1[curPos] = short(d_rand_org1[curPos] * (nPopulationNum - 1));

		//�������һ���ӽ���
		d_pRecombPos0[curPos] = d_rand_Tx[curPos] * (d_organLen[d_pOrgIdx0[curPos]] - 1);
	}
}

template <int BLOCK_SIZE> __global__ void
MSAGA_RECOMB_VERTICAL_RECOMB_POS(   short* d_pRecombPos0, short* d_pRecombPos1, short* d_pOrgIdx0, short* d_pOrgIdx1, short *d_population, float *d_rand_organ_idx, float* d_rand_Tx,   float* d_rand_org0, float* d_rand_org1, float fRecombVertical, int nPopulation, int nBlockWidth, int iGeneSpace, unsigned int *d_organLen, int nWidth, int nRecomb)
{
	// Block index
    int bx = blockIdx.x;
//    int by = blockIdx.y;
	
	// Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	int curPos = bx*blockDim.x+ty*nWidth+tx;
	int curPopulation = curPos%nRecomb;//��ǰλ������Ӧ�ĸ�����
	int curSeqNo = curPos/nRecomb;//��ǰλ������Ӧ�ĸ����еĵ�ǰ����

	d_pOrgIdx0[curPopulation]=-1;
	d_pOrgIdx1[curPopulation]=-1;
	d_pRecombPos0[curSeqNo*nRecomb+curPopulation]=-100;
	d_pRecombPos1[curSeqNo*nRecomb+curPopulation]=-100;
	
	if(d_rand_organ_idx[curPopulation]<fRecombVertical)//ÿ��tx��Ӧһ����ż��ŵĸ���
	{
		d_pOrgIdx0[curPopulation] = d_rand_org0[curPopulation]*(nPopulation-1);//�������һ�Խ��д�ֱ�ӽ��ĸ���
		d_pOrgIdx1[curPopulation] = d_rand_org1[curPopulation]*(nPopulation-1);
		int posOrg0 = d_rand_Tx[curPopulation]*(d_organLen[curPopulation]-1);//�������һ���ӽ���
		//ͳ�ƴӵ�һ�����忪ͷ���ӽ���֮ǰ�Ĳл�����
		int n=0;
		for( int i=0; i<=posOrg0; ++i )
		{
			if( d_population[d_pOrgIdx0[curPopulation]*nBlockWidth+curSeqNo*(nPopulation*nBlockWidth)+i]!=iGeneSpace )
			{
				++n;
			}
		}

		//���ҵڶ���������Ӧ�ӽ���λ��
		int j=0;
		int posOrg1=0;
		if( d_population[d_pOrgIdx0[curPopulation]*nBlockWidth+curSeqNo*(nPopulation*nBlockWidth)+0]==iGeneSpace && n==0 )//�ӵ�һ�����嵽�ӽ��㴦û�вл���ȫΪ�ո���posOrg1Ϊ�ڶ��������е�һ���л�ǰһλ��
		{
			while( d_population[d_pOrgIdx1[curPopulation]*nBlockWidth+curSeqNo*(nPopulation*nBlockWidth)+j]==iGeneSpace  )
			{
				++j;
			}
			posOrg1=j-1;
		}
		else
		{
			while( j<n )
			{
				if( d_population[d_pOrgIdx1[curPopulation]*nBlockWidth+curSeqNo*(nPopulation*nBlockWidth)+posOrg1]!=iGeneSpace )
				{
					++j;
				}
				++posOrg1;
			}
			if(posOrg1!=0)
			{
				--posOrg1;
			}
		}
		
		d_pRecombPos0[curSeqNo*nRecomb+curPopulation] = posOrg0;	
		d_pRecombPos1[curSeqNo*nRecomb+curPopulation] = posOrg1;
	}
}

__global__	void	MSAGA_RECOMB_HORIZEN_UNLIMIT(short *d_populationNextGen, short *d_population, float *d_rand_organ_idx, float* d_rand_Ty, float* d_rand_org0, float* d_rand_org1, float fRecombHorizen, int nOrgWidth, float rnOrgWidth, int nPopulationWidth, float rnPopulationWidth, int nWidth, int nPopulation, int nOrgNum, int nCellNum)
{
	// Block index
	int bx = blockIdx.x;
	//    int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int curPos = bx*blockDim.x+ty*nWidth+tx;
	if (curPos >= nCellNum)   return;
#if 0
	int orgIdx = (curPos%nPopulationWidth)/nOrgWidth;
	int seqIdx = curPos/nPopulationWidth;
	int posIdx = curPos-seqIdx*nPopulationWidth-orgIdx*nOrgWidth;
#endif
	int seqIdx = floor(curPos*rnPopulationWidth);
	int i = curPos - seqIdx*nPopulationWidth;
	int orgIdx = i*rnOrgWidth;
	int posIdx = curPos - seqIdx*nPopulationWidth - orgIdx*nOrgWidth;

	if(orgIdx==0) 
	{//���Ÿ���ֱ�ӽ�����һ��
		d_populationNextGen[curPos] = d_population[curPos];
		return;
	}		

	//result = 

	if( (nPopulation-1)%2!=0 && orgIdx==nPopulation-1 )//����������������Ϊ�����������һ������ֱ�Ӹ���
	{
		d_populationNextGen[curPos] = d_population[curPos];
		return;
	}

	int  curOrder=orgIdx;
	if( orgIdx%2==0 )//�����ǰ������ż����Ÿ���
	{
		curOrder=orgIdx-1;//ʹ����������������Ž��м���
	}

	int curIdx = curOrder/2;//half of the population size
	if(d_rand_organ_idx[curIdx]<fRecombHorizen)//������ˮƽ�ӽ������ĸ���
	{
		int bxOrg0 = d_rand_org0[curIdx]*(nPopulation-1);
		int bxOrg1 = d_rand_org1[curIdx]*(nPopulation-1);
		int selectSeqIdx = d_rand_Ty[curIdx]*(nOrgNum-1);

		if( orgIdx%2==0 )//�����ǰ������ż����Ÿ���
		{
			if(seqIdx==selectSeqIdx)
			{//exchange the cur Seq
				d_populationNextGen[curPos] = d_population[bxOrg0*nOrgWidth+selectSeqIdx*nPopulationWidth+posIdx];
			}
			else
			{
				d_populationNextGen[curPos] = d_population[bxOrg1*nOrgWidth+seqIdx*nPopulationWidth+posIdx];
			}
		}
		else//�����ǰ������������Ÿ���
		{
			if(seqIdx==selectSeqIdx)
			{//exchange the cur Seq
				d_populationNextGen[curPos] = d_population[bxOrg1*nOrgWidth+selectSeqIdx*nPopulationWidth+posIdx];
			}
			else
			{
				d_populationNextGen[curPos] = d_population[bxOrg0*nOrgWidth+seqIdx*nPopulationWidth+posIdx];
			}
		}
	}
	else
	{//�������ӽ������ĸ���ֱ�ӿ���
		d_populationNextGen[curPos] = d_population[curPos];
	}
}

template <int BLOCK_SIZE> __global__ void
MSAGA_RECOMB_HORIZEN(   short *d_populationNextGen, short *d_population, float *d_rand_organ_idx, float* d_rand_Ty, float* d_rand_org0, float* d_rand_org1, float fRecombHorizen, int organNum, int nPopulation )
{
	// Block index
    int bx = blockIdx.x;
//    int by = blockIdx.y;
	
	// Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	int nBlockWidth = blockDim.x;
	int curPos=bx*nBlockWidth+ty*(nPopulation*nBlockWidth)+tx;
	if(bx==0) 
	{//���Ÿ���ֱ�ӽ�����һ��
		d_populationNextGen[curPos] = d_population[curPos];
		return;
	}	

	if( (nPopulation-1)%2!=0 && bx==nPopulation-1 )//����������������Ϊ�����������һ������ֱ�Ӹ���
	{
		d_populationNextGen[curPos] = d_population[curPos];
		return;
	}

	int  curOrder=bx;
	if( bx%2==0 )//�����ǰ������ż����Ÿ���
	{
		curOrder=bx-1;//ʹ����������������Ž��м���
	}

	int curIdx = curOrder/2;//half of the population size
	if(d_rand_organ_idx[curIdx]<fRecombHorizen)//������ˮƽ�ӽ������ĸ���
	{
		int bxOrg0 = d_rand_org0[curIdx]*(nPopulation-1);
		int bxOrg1 = d_rand_org1[curIdx]*(nPopulation-1);
		int selectTy = d_rand_Ty[curIdx]*(organNum-1);

		if( bx%2==0 )//�����ǰ������ż����Ÿ���
		{
			if(ty==selectTy)
			{//exchange the cur Seq
				d_populationNextGen[curPos] = d_population[bxOrg0*nBlockWidth+selectTy*(nPopulation*nBlockWidth)+tx];
			}
			else
			{
				d_populationNextGen[curPos] = d_population[bxOrg1*nBlockWidth+ty*(nPopulation*nBlockWidth)+tx];
			}
		}
		else//�����ǰ������������Ÿ���
		{
			if(ty==selectTy)
			{//exchange the cur Seq
				d_populationNextGen[curPos] = d_population[bxOrg1*nBlockWidth+selectTy*(nPopulation*nBlockWidth)+tx];
			}
			else
			{
				d_populationNextGen[curPos] = d_population[bxOrg0*nBlockWidth+ty*(nPopulation*nBlockWidth)+tx];
			}
		}
	}
	else
	{//�������ӽ������ĸ���ֱ�ӿ���
		d_populationNextGen[curPos] = d_population[curPos];
	}
}

__global__ void MSAGA_GAP_REDUCE_UNLIMIT(  short *d_populationNextGen, short *d_population, float *d_rand_organ_idx,  float* d_rand_Ty, short* d_gapPos_Tx, int nSeqNum, int nWidth, float fReduceGapChance, int GAPCODE, int nOrgWidth, float rnOrgWidth, int nPopulationWidth, float rnPopulationWidth, int nCellNum )
{
	// Block index
	int bx = blockIdx.x;
	//    int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int curPos = bx*blockDim.x+ty*nWidth+tx;
	if(curPos>=nCellNum)  return;
#if 0
	int nPopulationWidth = nOrgWidth*nPopulation;
	int orgIdx = curPos%nPopulationWidth/nOrgWidth;
	if (curPos >= nPopulation)  return;
	int seqIdx = curPos/nPopulationWidth;
	int posIdx = curPos-seqIdx*nPopulationWidth-orgIdx*nOrgWidth;
#endif
	int seqIdx = floor(curPos*rnPopulationWidth);
	int i = curPos - seqIdx*nPopulationWidth;
	int orgIdx = i*rnOrgWidth;
	int posIdx = curPos - seqIdx*nPopulationWidth - orgIdx*nOrgWidth;

	if(orgIdx==0) 
	{//���Ÿ���ֱ�ӽ�����һ��
		d_populationNextGen[curPos] = d_population[curPos];
		return;
	}	

	if(d_rand_organ_idx[orgIdx]<fReduceGapChance)
	{
		int reduceTy = d_rand_Ty[orgIdx]*(nSeqNum-1);
		if( seqIdx==reduceTy )
		{
			int reduceTx = d_gapPos_Tx[orgIdx];
			if( reduceTx!=-1 )//���пո�����ڼ���ʱ
			{
				if( posIdx<reduceTx )
				{//direct copy
					d_populationNextGen[curPos] = d_population[curPos];
				}
				else if( posIdx<(nOrgWidth-1) )
				{
					d_populationNextGen[curPos] = d_population[curPos+1];
				}
				else//���һ�����ݵ�ֱ����Ϊδʹ�õĿռ�Ǻ�
				{
					d_populationNextGen[curPos] = MSA_GA_CUDA_SPACE;
				}
			}
			else
			{
				d_populationNextGen[curPos] = d_population[curPos];
			}
		}
		else
		{//just copy to next generation
			d_populationNextGen[curPos] = d_population[curPos];
		}
	}
	else
	{//just copy to next generation
		d_populationNextGen[curPos] = d_population[curPos];
	}

}

template <int BLOCK_SIZE> __global__ void
MSAGA_GAP_REDUCE(  short *d_populationNextGen, short *d_population, float *d_rand_organ_idx,  float* d_rand_Ty, short* d_gapPos_Tx, int nSeqNum, int nWidth, float fReduceGapChance, int GAPCODE )
{
	// Block index
    int bx = blockIdx.x;
//    int by = blockIdx.y;
	
	// Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	int curPos=bx*blockDim.x+ty*nWidth+tx;
	if(bx==0) 
	{//���Ÿ���ֱ�ӽ�����һ��
		d_populationNextGen[curPos] = d_population[curPos];
		return;
	}	

	if(d_rand_organ_idx[bx]<fReduceGapChance)
	{
		int reduceTy = d_rand_Ty[bx]*(nSeqNum-1);
		if( ty==reduceTy )
		{
			int reduceTx = d_gapPos_Tx[bx];
			if( reduceTx!=-1 )//���пո�����ڼ���ʱ
			{
				if( tx<reduceTx )
				{//direct copy
					d_populationNextGen[curPos] = d_population[curPos];
				}
				else if( tx<(blockDim.x-1) )
				{
					d_populationNextGen[curPos] = d_population[curPos+1];
				}
				else//���һ�����ݵ�ֱ����Ϊδʹ�õĿռ�Ǻ�
				{
					d_populationNextGen[curPos] = MSA_GA_CUDA_SPACE;
				}
			}
			else
			{
				d_populationNextGen[curPos] = d_population[curPos];
			}
		}
		else
		{//just copy to next generation
			d_populationNextGen[curPos] = d_population[curPos];
		}
	}
	else
	{//just copy to next generation
		d_populationNextGen[curPos] = d_population[curPos];
	}
}

__global__ void MSAGA_GAP_INSERT_UNLIMIT(short *d_populationNextGen, short *d_population, float *d_rand_organ_idx, float* d_rand_Ty, float* d_rand_Tx, int populationNum, int nSeq, int nBlockWidth, float fOpenGapChance, int GAPCODE, int nWidth, int nOrgWidth, float rnOrgWidth, int nPopulationWidth, float rnPopulationWidth, unsigned int* d_OrgLen, int nCellNum)
{
	// Block index
	int bx = blockIdx.x;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int curPos = bx*blockDim.x + ty*nWidth + tx;
	if (curPos >= nCellNum)  return;
#if 0
	int orgIdx = curPos%nPopulationWidth/nOrgWidth;
	int seqIdx = curPos/nPopulationWidth;
	int posIdx = curPos-seqIdx*nPopulationWidth-orgIdx*nOrgWidth;
	int curPopulationPos = orgIdx*nOrgWidth+seqIdx*nPopulationWidth+posIdx;
#endif
	int seqIdx = floor(curPos*rnPopulationWidth);
	int i = curPos - seqIdx*nPopulationWidth;
	int orgIdx = i*rnOrgWidth;
	int posIdx = curPos - seqIdx*nPopulationWidth - orgIdx*nOrgWidth;

	if(orgIdx==0) 
	{//���Ÿ���ֱ�ӽ�����һ��
		d_populationNextGen[curPos] = d_population[curPos];
		return;
	}	

	if(d_rand_organ_idx[orgIdx]<fOpenGapChance)
	{//open gap
		int insertSeqIdx = d_rand_Ty[orgIdx]*(nSeq-1);
		if( seqIdx==insertSeqIdx )//��ǰ����Ϊ��Ҫ����GAP������
		{
			int insertPosIdx = d_rand_Tx[orgIdx]*(d_OrgLen[orgIdx]-1);
			if(  insertPosIdx==0 )//������λ��������ͷ��
			{
				if( posIdx==insertPosIdx )
				{//ֱ������һ��������ͷ������GAP
					d_populationNextGen[curPos] = GAPCODE;
				}
				else//����һ���У�ԭ���������л��������һλ
				{
					d_populationNextGen[curPos] = d_population[curPos - 1];
				}
			}
			else//������λ�ò�������ͷ��
			{
				if( posIdx<insertPosIdx )
				{//��ǰλ��λ�ڲ����ǰ������һ��ֱ�Ӹ�����һ����Ӧ�л���
					d_populationNextGen[curPos] = d_population[curPos];
				}
				else if( posIdx==insertPosIdx )
				{//��ǰλ��Ϊ����㣬�����GAP
					d_populationNextGen[curPos] = GAPCODE;
				}
				else
				{//��ǰλ���ڲ�����������һ���У�ԭ������Ӧ�л���������ƶ�һλ��
					d_populationNextGen[curPos] = d_population[curPos - 1];
				}
			}
		}
		else//��ǰ���в���Ҫ����GAP�����У���ֱ�Ӹ���.
		{
			d_populationNextGen[curPos] = d_population[curPos];
		}
	}
	else//��ǰ�������岻��ҪGapInsertMutation����ֱ�Ӹ��ơ�
	{
		d_populationNextGen[curPos] = d_population[curPos];
	}
}

template <int BLOCK_SIZE> __global__ void
MSAGA_GAP_INSERT(  short *d_populationNextGen, short *d_population, float *d_rand_organ_idx,  float* d_rand_Ty, float* d_rand_Tx, int populationNum, int nSeq, int nBlockWidth, float fOpenGapChance, int GAPCODE )
{
	// Block index
    int bx = blockIdx.x;
// int by = blockIdx.y;
	
	// Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	int curPos=bx*nBlockWidth+ty*(populationNum*nBlockWidth)+tx;
	if(bx==0) 
	{//���Ÿ���ֱ�ӽ�����һ��
		d_populationNextGen[curPos] = d_population[curPos];
		return;
	}	

	if(d_rand_organ_idx[bx]<fOpenGapChance)
	{//open gap
		int insertTy = d_rand_Ty[bx]*(nSeq-1);
		if( ty==insertTy )//��ǰ����Ϊ��Ҫ����GAP������
		{
			int insertTx = d_rand_Tx[bx]*(nBlockWidth-1);
			if(d_population[bx*nBlockWidth+ty*(populationNum*nBlockWidth)+insertTx]==MSA_GA_CUDA_SPACE)
			{//�������λ�ó�����Ч��������������insertTx=(nBlockWidth-1)-insertTxΪ�µĲ���λ�ã���Ϊ�����Ŀռ�ֻռȫ���ռ��20%���ң���˵ڶ���Tx��Ȼָ�������ݵ�����
				insertTx = nBlockWidth-1-insertTx;
			}
			if(  insertTx==0 )//������λ��������ͷ��
			{
				if( tx==insertTx )
				{//ֱ������һ��������ͷ������GAP
					d_populationNextGen[curPos] = GAPCODE;
				}
				else//����һ���У�ԭ���������л��������һλ
				{
					d_populationNextGen[curPos] = d_population[curPos-1];
				}
			}
			else//������λ�ò�������ͷ��
			{
				if( tx<insertTx )
				{//��ǰλ��λ�ڲ����ǰ������һ��ֱ�Ӹ�����һ����Ӧ�л���
					d_populationNextGen[curPos] = d_population[curPos];
				}
				else if( tx==insertTx )
				{//��ǰλ��Ϊ����㣬�����GAP
					d_populationNextGen[curPos] = GAPCODE;
				}
				else
				{//��ǰλ���ڲ�����������һ���У�ԭ������Ӧ�л���������ƶ�һλ��
					d_populationNextGen[curPos] = d_population[curPos-1];
				}
			}
		}
		else//��ǰ���в���Ҫ����GAP�����У���ֱ�Ӹ���.
		{
			d_populationNextGen[curPos] = d_population[curPos];
		}
	}
	else//��ǰ�������岻��ҪGapInsertMutation����ֱ�Ӹ��ơ�
	{
		d_populationNextGen[curPos] = d_population[curPos];
	}
}

__global__	void	MSAGA_GAP_EXTENSION_UNLIMIT(short *d_populationNextGen, short *d_population, float *d_rand_organ_idx,  float* d_rand_Ty, short* d_gapExtendPos_Tx, int populationNum, int nSeq, float fOpenGapChance, int GAPCODE, int nWidth, int nOrgWidth, float rnOrgWidth, int nPopulationWidth, float rnPopulationWidth, int nCellNum )
{
	// Block index
    int bx = blockIdx.x;
// int by = blockIdx.y;
	
	// Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	int curPos = bx*blockDim.x+ty*nWidth+tx;
	if(curPos>=nCellNum)  return;
#if 0
	int orgIdx = curPos%nPopulationWidth/nOrgWidth;
	if (orgIdx >= populationNum)  return;
	int seqIdx = curPos/nPopulationWidth;
	int posIdx = curPos-seqIdx*nPopulationWidth-orgIdx*nOrgWidth;
#endif
	int seqIdx = floor(curPos*rnPopulationWidth);
	int i = curPos - seqIdx*nPopulationWidth;
	int orgIdx = i*rnOrgWidth;
	int posIdx = curPos - seqIdx*nPopulationWidth - orgIdx*nOrgWidth;

	if(orgIdx==0) 
	{//���Ÿ���ֱ�ӽ�����һ��
		d_populationNextGen[curPos] = d_population[curPos];
		return;
	}	

	if(d_rand_organ_idx[orgIdx]<fOpenGapChance)
	{//open gap
		int insertTy = d_rand_Ty[orgIdx]*(nSeq-1);
		if( seqIdx==insertTy )//��ǰ����Ϊ��Ҫ����GAP������
		{
			int insertTx = d_gapExtendPos_Tx[orgIdx];
			if(  insertTx==0 )//������λ��������ͷ��
			{
				if( posIdx==insertTx )
				{//ֱ������һ��������ͷ������GAP
					d_populationNextGen[curPos] = GAPCODE;
				}
				else//����һ���У�ԭ���������л��������һλ
				{
					d_populationNextGen[curPos] = d_population[curPos-1];
				}
			}
			else//������λ�ò�������ͷ��
			{

				if( posIdx<insertTx )
				{//��ǰλ��λ�ڲ����ǰ������һ��ֱ�Ӹ�����һ����Ӧ�л���
					d_populationNextGen[curPos] = d_population[curPos];
				}
				else if( posIdx==insertTx )
				{//��ǰλ��Ϊ����㣬�����GAP
					d_populationNextGen[curPos] = GAPCODE;
				}
				else
				{//��ǰλ���ڲ�����������һ���У�ԭ������Ӧ�л���������ƶ�һλ��
					d_populationNextGen[curPos] = d_population[curPos-1];
				}
			}
		}
		else//��ǰ���в���Ҫ����GAP�����У���ֱ�Ӹ���.
		{
			d_populationNextGen[curPos] = d_population[curPos];
		}
	}
	else//��ǰ�������岻��ҪGapInsertMutation����ֱ�Ӹ��ơ�
	{
		d_populationNextGen[curPos] = d_population[curPos];
	}
}

template <int BLOCK_SIZE> __global__ void
MSAGA_GAP_EXTENSION(  short *d_populationNextGen, short *d_population, float *d_rand_organ_idx,  float* d_rand_Ty, short* d_gapExtendPos_Tx, int populationNum, int nSeq, int nBlockWidth, float fOpenGapChance, int GAPCODE )
{
	// Block index
    int bx = blockIdx.x;
// int by = blockIdx.y;
	
	// Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	int curPos=bx*nBlockWidth+ty*(populationNum*nBlockWidth)+tx;
	if(bx==0) 
	{//���Ÿ���ֱ�ӽ�����һ��
		d_populationNextGen[curPos] = d_population[curPos];
		return;
	}	

	if(d_rand_organ_idx[bx]<fOpenGapChance)
	{//open gap
		int insertTy = d_rand_Ty[bx]*(nSeq-1);
		if( ty==insertTy )//��ǰ����Ϊ��Ҫ����GAP������
		{
			int insertTx = d_gapExtendPos_Tx[bx];
			//if(d_population[bx*nBlockWidth+ty*(populationNum*nBlockWidth)+insertTx]==MSA_GA_CUDA_SPACE)
			//{//�������λ�ó�����Ч��������������insertTx=(nBlockWidth-1)-insertTxΪ�µĲ���λ�ã���Ϊ�����Ŀռ�ֻռȫ���ռ��20%���ң���˵ڶ���Tx��Ȼָ�������ݵ�����
			//	insertTx = nBlockWidth-1-insertTx;
			//}
			if(  insertTx==0 )//������λ��������ͷ��
			{
				if( tx==insertTx )
				{//ֱ������һ��������ͷ������GAP
					d_populationNextGen[curPos] = GAPCODE;
				}
				else//����һ���У�ԭ���������л��������һλ
				{
					d_populationNextGen[curPos] = d_population[curPos-1];
				}
			}
			else//������λ�ò�������ͷ��
			{

				if( tx<insertTx )
				{//��ǰλ��λ�ڲ����ǰ������һ��ֱ�Ӹ�����һ����Ӧ�л���
					d_populationNextGen[curPos] = d_population[curPos];
				}
				else if( tx==insertTx )
				{//��ǰλ��Ϊ����㣬�����GAP
					d_populationNextGen[curPos] = GAPCODE;
				}
				else
				{//��ǰλ���ڲ�����������һ���У�ԭ������Ӧ�л���������ƶ�һλ��
					d_populationNextGen[curPos] = d_population[curPos-1];
				}
			}
		}
		else//��ǰ���в���Ҫ����GAP�����У���ֱ�Ӹ���.
		{
			d_populationNextGen[curPos] = d_population[curPos];
		}
	}
	else//��ǰ�������岻��ҪGapInsertMutation����ֱ�Ӹ��ơ�
	{
		d_populationNextGen[curPos] = d_population[curPos];
	}
}

__global__  void  MSAGA_SELECT_NEXT_GEN_UNLIMIT(short *d_populationNextGen, short *d_population, uint *d_organ_index, int nOrgWidth, float rnOrgWidth, int nPopulationWidth, float rnPopulationWidth, int nWidth, int nCellNum)
{
	// Block index
    int bx = blockIdx.x;
	
	// Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	int curPos = bx*blockDim.x+ty*nWidth+tx;
	if (curPos >= nCellNum)  return;
	//curPos = nPopulationWidth*j+i
//#if 0
	int j = floor(curPos*rnPopulationWidth);
	int i = curPos - j*nPopulationWidth;
	int destOrgIdx = i*rnOrgWidth;
//#endif

//	int destOrgIdx = curPos%nPopulationWidth / nOrgWidth;

	int srcOrgIdx = d_organ_index[destOrgIdx];
#if 0
	int seqIdx = curPos/nPopulationWidth;
	int posIdx = curPos - seqIdx*nPopulationWidth - destOrgIdx*nOrgWidth;
	d_populationNextGen[curPos] = d_population[srcOrgIdx*nOrgWidth + seqIdx*nPopulationWidth + posIdx];
#endif
	d_populationNextGen[curPos] = d_population[srcOrgIdx*nOrgWidth + curPos - destOrgIdx*nOrgWidth];
}


template <int BLOCK_SIZE> __global__ void
MSAGA_SELECT_NEXT_GEN( short *d_populationNextGen, short *d_population, uint *d_organ_index,  int nWidth)
{
	// Block index
    int bx = blockIdx.x;
	
	// Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	int offsetBlock = d_organ_index[bx];
	d_populationNextGen[bx*blockDim.x+ty*nWidth+tx] = d_population[offsetBlock*blockDim.x+ty*nWidth+tx];
}

template <int BLOCK_SIZE> __global__ void
MSAGA_SELECT_MAX_CUDA(uint *d_selected_organ_idx, uint* d_organ_idx, short* d_organ_score, float* d_random_organ_idx, short organ_num, short candidateLen)
{
	// Block index
    int bx = blockIdx.x;
//    int by = blockIdx.y;
	
	// Thread index
    int tx = threadIdx.x;
//    int ty = threadIdx.y;

	int curPos = bx*blockDim.x+tx;

	if( curPos==0 )
	{
		d_selected_organ_idx[curPos] = d_organ_idx[0];//the best one directly selected into next generation.
		return;
	}

	int curIdxPos = curPos*candidateLen;
	short IdxCandidate= d_organ_idx[(int)(d_random_organ_idx[curIdxPos]*(organ_num-1))];
	short maxScore = d_organ_score[IdxCandidate];
	for( int i=1; i<candidateLen; ++i )
	{
		short idx = d_organ_idx[(int)d_random_organ_idx[curIdxPos+i]*(organ_num-1)];
		if(maxScore<d_organ_score[idx])
		{
			maxScore = d_organ_score[idx];
			IdxCandidate = idx;
		}
	}
	d_selected_organ_idx[curPos] = IdxCandidate;
}

__global__ void MSAGA_SELECT_MAX_CUDA_UNLIMIT(uint *d_selected_organ_idx, uint* d_organ_idx, float* d_organ_score, float* d_random_organ_idx, int organ_num, short candidateLen, int nWidth)
{
	// Block index
	int bx = blockIdx.x;
	//    int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int curOrgIdx = bx*blockDim.x + ty*nWidth + tx;
	if (curOrgIdx >= organ_num)  return;
	//int orgIdx = curPos%nPopulationWidth / nOrgWidth;
	if (curOrgIdx == 0)
	{
		d_selected_organ_idx[curOrgIdx] = d_organ_idx[0];//the best one directly selected into next generation.
		return;
	}

	int curIdxPos = curOrgIdx*candidateLen;
	short IdxCandidate = d_organ_idx[(int)(d_random_organ_idx[curIdxPos] * (organ_num - 1))];
	short maxScore = d_organ_score[IdxCandidate];
	for (int i = 1; i<candidateLen; ++i)
	{
		short idx = d_organ_idx[(int)d_random_organ_idx[curIdxPos + i] * (organ_num - 1)];
		if (maxScore<d_organ_score[idx])
		{
			maxScore = d_organ_score[idx];
			IdxCandidate = idx;
		}
	}
	d_selected_organ_idx[curOrgIdx] = IdxCandidate;
}

__global__ void MSAGA_RECORD_NEXT_GEN_LEN_UNLIMIT(unsigned int* d_orgLenNextGen, unsigned int* d_orgLen, uint* d_SelectedOrgIdx, short nWidth, int nCellNum)
{
	// Block index
	int bx = blockIdx.x;
	//    int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int curOrgIdx = bx*blockDim.x + ty*nWidth + tx;
	if (curOrgIdx >= nCellNum)  return;
	d_orgLenNextGen[curOrgIdx] = d_orgLen[d_SelectedOrgIdx[curOrgIdx]];
}

void		CMSAGA_CUDA_Algorithm::CUDA_DEV2HOST( void *h_mem, const void * const d_mem,  unsigned int size_mem )
{
	 // copy device memory to host
	hipError_t error;
    error = hipMemcpy(h_mem, d_mem, size_mem, hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
		throw   CAppException( DEF_EXCEPTION_UNEXPECTED,DEF_EXCEPTION_LEVEL_EXIT_TRY_BLOCK
			,__EXCEPTION_SITE__ ,"hipMemcpy  Device2Host Failured !" );
    }
}
 
void		CMSAGA_CUDA_Algorithm::CUDA_MALLOC( void **d_mem, unsigned int size_mem )
{
	hipError_t error;
	error = hipMalloc(d_mem, size_mem);
	if (error != hipSuccess)
	{
		throw   CAppException( DEF_EXCEPTION_UNEXPECTED,DEF_EXCEPTION_LEVEL_EXIT_TRY_BLOCK
			,__EXCEPTION_SITE__ ,"hipMalloc Failured !" );
	}
}

void		CMSAGA_CUDA_Algorithm::CUDA_MALLOC( void **d_mem, void *h_mem, unsigned int size_mem )
{
	hipError_t error;
	error = hipMalloc(d_mem, size_mem);
	if (error != hipSuccess)
	{
		throw   CAppException( DEF_EXCEPTION_UNEXPECTED,DEF_EXCEPTION_LEVEL_EXIT_TRY_BLOCK
			,__EXCEPTION_SITE__ ,"hipMalloc Failured !" );
	}
	error = hipMemcpy(*d_mem, h_mem, size_mem, hipMemcpyHostToDevice);
	if (error != hipSuccess)
	{
		throw   CAppException( DEF_EXCEPTION_UNEXPECTED,DEF_EXCEPTION_LEVEL_EXIT_TRY_BLOCK
			,__EXCEPTION_SITE__ ,"hipMemcpy  Host2Device Failured!" );
	}
}

void		CMSAGA_CUDA_Algorithm::CUDA_FREE( void *d_mem )
{
	hipError_t error;
	error = hipFree(d_mem);
	if (error != hipSuccess)
	{
		throw   CAppException( DEF_EXCEPTION_UNEXPECTED,DEF_EXCEPTION_LEVEL_EXIT_TRY_BLOCK
			,__EXCEPTION_SITE__ ,"hipFree Failured !" );
	}
}

void	CMSAGA_CUDA_Algorithm::SetSubstitutionMat2GPU(  SubMatrixType type  )
{
	if( NULL!=m_pSubMatDevice )
	{
		if( type != m_curSubMatType )
		{
			hipFree( m_pSubMatDevice );
			CUDA_MALLOC( (void**)&m_pSubMatDevice, CGlobalSpace::m_sAlignParams.getSubMatrix(type), sizeof(short)*CGlobalSpace::m_sAlignParams.getSubMatrixSize(type) );
		}
	}
	else
	{
		m_curSubMatType = type;
		CUDA_MALLOC( (void**)&m_pSubMatDevice, CGlobalSpace::m_sAlignParams.getSubMatrix(type), sizeof(short)*CGlobalSpace::m_sAlignParams.getSubMatrixSize(type) );
	}
}

void		CMSAGA_CUDA_Algorithm::MutationGapReductionDeviceUnlimit()
{
	RandomNumberGeneratorDevice( m_pMutation_RandOrgan_Device, MSA_GA_POPULATION_SIZE );
	RandomNumberGeneratorDevice( m_pMutation_RandTy_Device, MSA_GA_POPULATION_SIZE );
	RandomNumberGeneratorDevice( m_pMutation_RandTx_Device, MSA_GA_POPULATION_SIZE );

	//mark the gap start position with 1, or else 0.
	dim3 thread( MSA_GA_CUDA_BLOCK_SIZE, MSA_GA_CUDA_BLOCK_SIZE );
	dim3 grid( ceil( (float)m_nCellNum/(thread.x*thread.y) ), 1 );
	MSAGA_MARK_GAP_START_UNLIMIT<<< grid, thread>>>( m_pGapStartRecorder_Device, m_pPopulationDevice[m_curPopulationIndex], 
													m_pMutation_RandOrgan_Device, m_pMutation_RandTy_Device, 
													m_pPopulation->nSeqSize, thread.x*grid.x, MSA_GA_EXTENSION_MUTATION, 
													CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE), 
													MSA_GA_CUDA_MAX_LEN, R_MSA_GA_CUDA_MAX_LEN, POPULATION_WIDTH, R_POPULATION_WIDTH, m_nCellNum);
	//dbgWriteFile( "gapStartRecorder.txt", m_pGapStartRecorder_Device, MSA_GA_POPULATION_SIZE*MSA_GA_CUDA_MAX_LEN, MSA_GA_CUDA_MAX_LEN  );
	grid.x = ceil((float)MSA_GA_POPULATION_SIZE / (thread.x*thread.y));
	MSAGA_LOCATE_GAP_UNLIMIT<32><<< grid, thread>>>( m_pGapPos_Tx_Device, m_pGapStartRecorder_Device, m_pMutation_RandTx_Device, m_pMutation_RandOrgan_Device,
													 MSA_GA_CUDA_MAX_LEN, grid.x*thread.x, MSA_GA_REDUCTION_MUTATION,
													 CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE), MSA_GA_POPULATION_SIZE);

	//delete one gap at Tx
	grid.x = ceil((float)m_nCellNum / (thread.x*thread.y));
	//dbgWriteFile( "gapPos_Tx.txt", m_pGapPos_Tx_Device, MSA_GA_POPULATION_SIZE, MSA_GA_POPULATION_SIZE  );
	MSAGA_GAP_REDUCE_UNLIMIT<<< grid, thread>>>( m_pPopulationDevice[1-m_curPopulationIndex], m_pPopulationDevice[m_curPopulationIndex], 
												m_pMutation_RandOrgan_Device, m_pMutation_RandTy_Device, m_pGapPos_Tx_Device,
												m_pPopulation->nSeqSize, thread.x*grid.x, MSA_GA_REDUCTION_MUTATION,
												CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE),  
												MSA_GA_CUDA_MAX_LEN, R_MSA_GA_CUDA_MAX_LEN, POPULATION_WIDTH, R_POPULATION_WIDTH, m_nCellNum);
	m_curPopulationIndex = 1-m_curPopulationIndex;

	//arrange the organs.
	ArrangeSeqDeviceUnlimit( m_pMutation_RandOrgan_Device, MSA_GA_REDUCTION_MUTATION );
}

void		CMSAGA_CUDA_Algorithm::MutationGapReductionDevice()
{
	RandomNumberGeneratorDevice( m_pMutation_RandOrgan_Device, MSA_GA_POPULATION_SIZE );
	RandomNumberGeneratorDevice( m_pMutation_RandTy_Device, MSA_GA_POPULATION_SIZE );
	RandomNumberGeneratorDevice( m_pMutation_RandTx_Device, MSA_GA_POPULATION_SIZE );
	
	//mark the gap start position with 1, or else 0.
	dim3 thread(MSA_GA_CUDA_MAX_LEN, m_pPopulation->nSeqSize );
	dim3 grid( MSA_GA_POPULATION_SIZE, 1 );
	MSAGA_MARK_GAP_START<32><<< grid, thread>>>( m_pGapStartRecorder_Device, m_pPopulationDevice[m_curPopulationIndex], 
		                                                                                        m_pMutation_RandOrgan_Device, m_pMutation_RandTy_Device, 
																								m_pPopulation->nSeqSize, MSA_GA_POPULATION_SIZE*MSA_GA_CUDA_MAX_LEN, MSA_GA_REDUCTION_MUTATION, 
																								CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE));

	thread.x = MSA_GA_POPULATION_SIZE;
	thread.y = 1;
	grid.x = 1;
	MSAGA_LOCATE_GAP<32><<< grid, thread>>>( m_pGapPos_Tx_Device, m_pGapStartRecorder_Device, m_pMutation_RandTx_Device, m_pMutation_RandOrgan_Device,
		                                                                               MSA_GA_CUDA_MAX_LEN, MSA_GA_POPULATION_SIZE*MSA_GA_CUDA_MAX_LEN, MSA_GA_REDUCTION_MUTATION,
																					   CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE));

	//delete one gap at Tx
	thread.x=MSA_GA_CUDA_MAX_LEN;
	thread.y=m_pPopulation->nSeqSize;
	grid.x=MSA_GA_POPULATION_SIZE;
	grid.y=1;
	MSAGA_GAP_REDUCE<32><<< grid, thread>>>( m_pPopulationDevice[1-m_curPopulationIndex], m_pPopulationDevice[m_curPopulationIndex], 
		                                                                               m_pMutation_RandOrgan_Device, m_pMutation_RandTy_Device, m_pGapPos_Tx_Device,
																					   m_pPopulation->nSeqSize, MSA_GA_POPULATION_SIZE*MSA_GA_CUDA_MAX_LEN, MSA_GA_REDUCTION_MUTATION,
																					   CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE));
	m_curPopulationIndex = 1-m_curPopulationIndex;

	//arrange the organs.
	ArrangeSeqDevice( m_pMutation_RandOrgan_Device, MSA_GA_REDUCTION_MUTATION );
}

void		CMSAGA_CUDA_Algorithm::MutationGapExtensionDeviceUnlimit()
{
	RandomNumberGeneratorDevice( m_pMutation_RandOrgan_Device, MSA_GA_POPULATION_SIZE );
	RandomNumberGeneratorDevice( m_pMutation_RandTy_Device, MSA_GA_POPULATION_SIZE );
	RandomNumberGeneratorDevice( m_pMutation_RandTx_Device, MSA_GA_POPULATION_SIZE );
	
	//mark the gap start position with 1, or else 0.
	dim3 thread( MSA_GA_CUDA_BLOCK_SIZE, MSA_GA_CUDA_BLOCK_SIZE );
	dim3 grid( ceil( (float)m_nCellNum/(thread.x*thread.y) ), 1 );
	MSAGA_MARK_GAP_START_UNLIMIT<<< grid, thread>>>( m_pGapStartRecorder_Device, m_pPopulationDevice[m_curPopulationIndex], 
													 m_pMutation_RandOrgan_Device, m_pMutation_RandTy_Device, 
													 m_pPopulation->nSeqSize, thread.x*grid.x, MSA_GA_EXTENSION_MUTATION, 
													 CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE), 
													 MSA_GA_CUDA_MAX_LEN, R_MSA_GA_CUDA_MAX_LEN, POPULATION_WIDTH, R_POPULATION_WIDTH, m_nCellNum);
	
	grid.x = ceil((float)MSA_GA_POPULATION_SIZE / (thread.x*thread.y));
	MSAGA_LOCATE_GAP_UNLIMIT<32><<< grid, thread>>>( m_pGapPos_Tx_Device, m_pGapStartRecorder_Device, m_pMutation_RandTx_Device, m_pMutation_RandOrgan_Device,
		                                             MSA_GA_CUDA_MAX_LEN, thread.x*grid.x, MSA_GA_EXTENSION_MUTATION,
													 CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE), MSA_GA_POPULATION_SIZE);
	
	//extend the gap at Tx
#if 0
	thread.x=MSA_GA_CUDA_BLOCK_SIZE;
	thread.y=thread.x;
	grid.x=ceil( (float)MSA_GA_CUDA_MAX_LEN*m_pPopulation->nSeqSize*MSA_GA_POPULATION_SIZE/(thread.x*thread.y) );
	grid.y=1;
#endif
	MSAGA_GAP_EXTENSION_UNLIMIT<<< grid, thread>>>( m_pPopulationDevice[1-m_curPopulationIndex], m_pPopulationDevice[m_curPopulationIndex], 
													m_pMutation_RandOrgan_Device, m_pMutation_RandTy_Device, m_pGapPos_Tx_Device,												
													MSA_GA_POPULATION_SIZE, m_OrganismSize, MSA_GA_EXTENSION_MUTATION, 												
													CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE), 												
													thread.x*grid.x, MSA_GA_CUDA_MAX_LEN, R_MSA_GA_CUDA_MAX_LEN,
													POPULATION_WIDTH, R_POPULATION_WIDTH, m_nCellNum);
	m_curPopulationIndex = 1-m_curPopulationIndex;
	
	//arrange the organs.
	ArrangeSeqDeviceUnlimit( m_pMutation_RandOrgan_Device, MSA_GA_EXTENSION_MUTATION );

}

void		CMSAGA_CUDA_Algorithm::MutationGapExtensionDevice()
{
	RandomNumberGeneratorDevice( m_pMutation_RandOrgan_Device, MSA_GA_POPULATION_SIZE );
	RandomNumberGeneratorDevice( m_pMutation_RandTy_Device, MSA_GA_POPULATION_SIZE );
	RandomNumberGeneratorDevice( m_pMutation_RandTx_Device, MSA_GA_POPULATION_SIZE );

	//mark the gap start position with 1, or else 0.
	dim3 thread(MSA_GA_CUDA_MAX_LEN, m_pPopulation->nSeqSize );
	dim3 grid( MSA_GA_POPULATION_SIZE, 1 );
	MSAGA_MARK_GAP_START<32><<< grid, thread>>>( m_pGapStartRecorder_Device, m_pPopulationDevice[m_curPopulationIndex], 
		                                                                                        m_pMutation_RandOrgan_Device, m_pMutation_RandTy_Device, 
																								m_pPopulation->nSeqSize, MSA_GA_POPULATION_SIZE*MSA_GA_CUDA_MAX_LEN, MSA_GA_EXTENSION_MUTATION, 
																								CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE));

	thread.x = MSA_GA_POPULATION_SIZE;
	thread.y = 1;
	grid.x = 1;
	MSAGA_LOCATE_GAP<32><<< grid, thread>>>( m_pGapPos_Tx_Device, m_pGapStartRecorder_Device, m_pMutation_RandTx_Device, m_pMutation_RandOrgan_Device,
		                                                                               MSA_GA_CUDA_MAX_LEN, MSA_GA_POPULATION_SIZE*MSA_GA_CUDA_MAX_LEN, MSA_GA_EXTENSION_MUTATION,
																					   CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE));

	//extend the gap at Tx
	thread.x=MSA_GA_CUDA_MAX_LEN;
	thread.y=m_pPopulation->nSeqSize;
	grid.x=MSA_GA_POPULATION_SIZE;
	grid.y=1;
	MSAGA_GAP_EXTENSION<32><<< grid, thread>>>( m_pPopulationDevice[1-m_curPopulationIndex], m_pPopulationDevice[m_curPopulationIndex], 
		                                                                                     m_pMutation_RandOrgan_Device, m_pMutation_RandTy_Device, m_pGapPos_Tx_Device,
																					         MSA_GA_POPULATION_SIZE, m_OrganismSize, MSA_GA_CUDA_MAX_LEN, MSA_GA_EXTENSION_MUTATION,
																					         CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE));
	m_curPopulationIndex = 1-m_curPopulationIndex;

	//arrange the organs.
	ArrangeSeqDevice( m_pMutation_RandOrgan_Device, MSA_GA_EXTENSION_MUTATION );
}

void		CMSAGA_CUDA_Algorithm::MutationGapInsertDeviceUnlimit()
{
	RandomNumberGeneratorDevice( m_pMutation_RandOrgan_Device, MSA_GA_POPULATION_SIZE );
	RandomNumberGeneratorDevice( m_pMutation_RandTx_Device, MSA_GA_POPULATION_SIZE );
	RandomNumberGeneratorDevice( m_pMutation_RandTy_Device, MSA_GA_POPULATION_SIZE );

	//locate the position of gap insert seq and position.
	dim3 thread( MSA_GA_CUDA_BLOCK_SIZE, MSA_GA_CUDA_BLOCK_SIZE );
	dim3 grid(ceil((float)m_nCellNum/(thread.x*thread.y)), 1);
	MSAGA_GAP_INSERT_UNLIMIT<<< grid, thread>>>( m_pPopulationDevice[1-m_curPopulationIndex], m_pPopulationDevice[m_curPopulationIndex], 
												 m_pMutation_RandOrgan_Device, m_pMutation_RandTy_Device, m_pMutation_RandTx_Device,
												 MSA_GA_POPULATION_SIZE, m_OrganismSize, MSA_GA_CUDA_MAX_LEN, MSA_GA_BLOCK_MUTATION,
												 CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE),
												 thread.x*grid.x, MSA_GA_CUDA_MAX_LEN, R_MSA_GA_CUDA_MAX_LEN, POPULATION_WIDTH, R_POPULATION_WIDTH,
												 m_pOrganLenDevice[m_curOrganLenIndex], m_nCellNum);
	m_curPopulationIndex = 1-m_curPopulationIndex;
	//dbgCheckSequence();
	//arrange the organs.
	ArrangeSeqDeviceUnlimit( m_pMutation_RandOrgan_Device, MSA_GA_BLOCK_MUTATION );
	//dbgCheckSequence();
}

void		CMSAGA_CUDA_Algorithm::MutationGapInsertDevice()
{
	//gap insert mutation
	RandomNumberGeneratorDevice( m_pMutation_RandOrgan_Device, MSA_GA_POPULATION_SIZE );
	RandomNumberGeneratorDevice( m_pMutation_RandTx_Device, MSA_GA_POPULATION_SIZE );
	RandomNumberGeneratorDevice( m_pMutation_RandTy_Device, MSA_GA_POPULATION_SIZE );

	//locate the position of gap insert seq and position.
	dim3 thread(MSA_GA_CUDA_MAX_LEN, m_pPopulation->nSeqSize );
	dim3 grid( MSA_GA_POPULATION_SIZE, 1 );
	MSAGA_GAP_INSERT<32><<< grid, thread>>>( m_pPopulationDevice[1-m_curPopulationIndex], m_pPopulationDevice[m_curPopulationIndex], 
		                                                                              m_pMutation_RandOrgan_Device, m_pMutation_RandTy_Device, m_pMutation_RandTx_Device,
																					  MSA_GA_POPULATION_SIZE, m_OrganismSize, MSA_GA_CUDA_MAX_LEN, MSA_GA_BLOCK_MUTATION,
																					  CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE));
	m_curPopulationIndex = 1-m_curPopulationIndex;


	//arrange the organs.
	ArrangeSeqDevice( m_pMutation_RandOrgan_Device, MSA_GA_BLOCK_MUTATION );
}

bool		CMSAGA_CUDA_Algorithm::FitnessDeviceUnlimit()
{
	dim3 thread(MSA_GA_CUDA_BLOCK_SIZE, MSA_GA_CUDA_BLOCK_SIZE);
	dim3 grid(ceil((float)MSA_GA_CUDA_MAX_LEN*m_pPopulation->nSeqSize*MSA_GA_POPULATION_SIZE / (thread.x*thread.y)), 1);
//	dbgWriteFile("score_before.txt", m_pOrganScore_Device, MSA_GA_POPULATION_SIZE, MSA_GA_POPULATION_SIZE);
//	dbgWriteSeqsFromDevice2File("before scoring.txt");
	//sp unit score
	MSAGA_SPS_UNIT_CUDA_UNLIMIT << < grid, thread >> >(m_pUnitScore_Device, m_pPopulationDevice[m_curPopulationIndex], m_pOrganLenDevice[m_curPopulationIndex], m_pSubMatDevice, m_pSeqWeight_Device,
													  m_pPopulation->nSeqSize, thread.x*grid.x, POPULATION_WIDTH, R_POPULATION_WIDTH, MSA_GA_CUDA_MAX_LEN, R_MSA_GA_CUDA_MAX_LEN,
													  CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE), m_rnPairs,
													  CGlobalSpace::m_sAlignParams.m_SubstitutionMatMgr.getGapOpenCost(MSAGA_MAT_TYPE), 
													  CGlobalSpace::m_sAlignParams.m_SubstitutionMatMgr.getGapExtendCost(MSAGA_MAT_TYPE));

//	dbgWriteFile(std::string("UnitScore.txt"), m_pUnitScore_Device, MSA_GA_CUDA_MAX_LEN*MSA_GA_POPULATION_SIZE*m_pPopulation->nSeqSize, MSA_GA_CUDA_MAX_LEN*MSA_GA_POPULATION_SIZE*m_pPopulation->nSeqSize);

	//column score
	grid.x = ceil((float)MSA_GA_CUDA_MAX_LEN*MSA_GA_POPULATION_SIZE / (thread.x*thread.y));
	MSAGA_SPS_COLSUM_CUDA_UNLIMIT << < grid, thread >> >(m_pColumnScore_Device, m_pUnitScore_Device, m_pPopulation->nSeqSize,
		thread.x*grid.x, MSA_GA_POPULATION_SIZE, POPULATION_WIDTH, MSA_GA_CUDA_MAX_LEN, R_MSA_GA_CUDA_MAX_LEN);

//	dbgWriteFile(std::string("ColScore.txt"), m_pUnitScore_Device, MSA_GA_CUDA_MAX_LEN*MSA_GA_POPULATION_SIZE*m_pPopulation->nSeqSize, MSA_GA_CUDA_MAX_LEN*MSA_GA_POPULATION_SIZE*m_pPopulation->nSeqSize);

	grid.x = ceil((float)MSA_GA_POPULATION_SIZE / (thread.x*thread.y));
	MSAGA_SPS_ORGAN_CUDA_UNLIMIT << < grid, thread >> >(m_pOrganScore_Device, m_pColumnScore_Device, thread.x*grid.x, MSA_GA_CUDA_MAX_LEN, MSA_GA_POPULATION_SIZE);

//	dbgWriteFile("score_after.txt", m_pOrganScore_Device, MSA_GA_POPULATION_SIZE, MSA_GA_POPULATION_SIZE);

	return false;
}

void		CMSAGA_CUDA_Algorithm::InitArrangement()
{
	CUDA_MALLOC( (void**)&m_pGapRecorder_Device,  sizeof(short)*MSA_GA_POPULATION_SIZE*MSA_GA_CUDA_MAX_LEN );
	CUDA_MALLOC( (void**)&m_pRangeRecorder_Device,  sizeof(short)*MSA_GA_POPULATION_SIZE*MSA_GA_CUDA_MAX_LEN );
	CUDA_MALLOC( (void**)&m_pGapLenRecorder_Device,  sizeof(short)*MSA_GA_POPULATION_SIZE*MSA_GA_CUDA_MAX_LEN );
}

void		CMSAGA_CUDA_Algorithm::InitSelection()
{
	float size=log10((float)MSA_GA_POPULATION_SIZE)/log10((float)2);
	m_nOrganSize_Align = pow(2.0f, ceil(size));

	unsigned int size_organ_score_align = m_nOrganSize_Align;//SHARED_SIZE_LIMIT;
	unsigned int mem_size_organ_score_align =  sizeof(float) * size_organ_score_align;
	unsigned int mem_size_organ_index_align = sizeof(uint) * size_organ_score_align;;

	CUDA_MALLOC( (void**)&m_pOrganScore_Align_Device,  mem_size_organ_score_align );
	CUDA_MALLOC( (void**)&m_pOrganIndex_Align_Device,  mem_size_organ_index_align );
	CUDA_MALLOC( (void**)&m_pOrganScore_Align_Sorted_Device,  mem_size_organ_score_align );
	CUDA_MALLOC( (void**)&m_pOrganIndex_Align_Sorted_Device,  mem_size_organ_index_align );

	//using tournament selection method to select the organ
	m_nRandom_organ_idx = sizeof(float)*MSA_GA_CUDA_SELECTION_CANDIDATE*MSA_GA_POPULATION_SIZE;
	CUDA_MALLOC( (void**)&m_pRandomOrganIdx_Device,  m_nRandom_organ_idx );

	CUDA_MALLOC( (void**)&m_pSelectedOrganIdx_Device,  sizeof(uint)*MSA_GA_POPULATION_SIZE );
}

void		CMSAGA_CUDA_Algorithm::InitFitness()
{
	//unit score
	unsigned int size_score_unit = m_pPopulation->nSeqSize*MSA_GA_CUDA_MAX_LEN*MSA_GA_POPULATION_SIZE;
	CUDA_MALLOC((void**)&m_pUnitScore_Device, sizeof(float)*size_score_unit);
	//load seq weight
	unsigned int size_seq_weight = size_score_unit;
	unsigned int mem_size_seq_weight = sizeof(float)*size_seq_weight;
	CUDA_MALLOC((void**)&m_pSeqWeight_Device, mem_size_seq_weight);
	float *pSeqWeight = new float[size_seq_weight];
	for (int i = 0; i < m_pPopulation->nSeqSize; ++i)
	{
		for (int j = 0; j < POPULATION_WIDTH; ++j)
		{
			pSeqWeight[i*POPULATION_WIDTH + j] = m_SeqWeight[i];
		}
	}

	CUDA_MALLOC((void**)&m_pSeqWeight_Device, pSeqWeight, mem_size_seq_weight);
	SAFE_DELETE_ARRAY(pSeqWeight);

	//column score
	unsigned int size_score_colsum = MSA_GA_CUDA_MAX_LEN*MSA_GA_POPULATION_SIZE;
	CUDA_MALLOC((void**)&m_pColumnScore_Device, sizeof(float)*size_score_colsum);
	//organ score
	unsigned int size_score_population = MSA_GA_POPULATION_SIZE;
	CUDA_MALLOC((void**)&m_pOrganScore_Device, sizeof(float)*size_score_population);
}

void		CMSAGA_CUDA_Algorithm::InitRecomb()
{
	int	nRecomb = (MSA_GA_POPULATION_SIZE-1)/2;//half of the population number

	CUDA_MALLOC( (void**)&m_pRecomb_RandOrgan_Device,  sizeof(float)*nRecomb );
	CUDA_MALLOC( (void**)&m_pRecomb_RandTy_Device,  sizeof(float)*nRecomb );

	//hipMemset( (void*)m_pRecomb_RandOrgan_Device, 1, sizeof(float)*nRecomb );
	//hipMemset( (void*)m_pRecomb_RandTy_Device, 2, sizeof(float)*nRecomb );
	//dbgWriteFile( "m_pRecomb_RandOrgan_Device.txt", m_pRecomb_RandOrgan_Device, nRecomb, nRecomb );
	//dbgWriteFile( "m_pRecomb_RandTy_Device.txt", m_pRecomb_RandTy_Device, nRecomb, nRecomb );

	CUDA_MALLOC( (void**)&m_pRecomb_RandOrganIdx0_Device,  sizeof(float)*nRecomb );
	CUDA_MALLOC( (void**)&m_pRecomb_RandOrganIdx1_Device,  sizeof(float)*nRecomb );

	//for vertical recombination
	CUDA_MALLOC( (void**)&m_pPos1_Min_Device,  sizeof(short)*nRecomb );
	CUDA_MALLOC( (void**)&m_pPos1_Max_Device,  sizeof(short)*nRecomb );

	CUDA_MALLOC( (void**)&m_pRecomb_RandTx_Device,  sizeof(float)*nRecomb );

	CUDA_MALLOC( (void**)&m_pOrganIndex0_Device,  sizeof(short)*nRecomb );
	CUDA_MALLOC( (void**)&m_pOrganIndex1_Device,  sizeof(short)*nRecomb );

	CUDA_MALLOC( (void**)&m_pVerticalRecomb_pos0_Device,  sizeof(short)*nRecomb );
	CUDA_MALLOC( (void**)&m_pVerticalRecomb_pos1_Device,  sizeof(short)*nRecomb*m_pPopulation->nSeqSize );
}

void		CMSAGA_CUDA_Algorithm::InitMutation()
{
	CUDA_MALLOC( (void**)&m_pMutation_RandOrgan_Device,  sizeof(float)*MSA_GA_POPULATION_SIZE );
	CUDA_MALLOC( (void**)&m_pMutation_RandTx_Device,  sizeof(float)*MSA_GA_POPULATION_SIZE );
	CUDA_MALLOC( (void**)&m_pMutation_RandTy_Device,  sizeof(float)*MSA_GA_POPULATION_SIZE );
	CUDA_MALLOC( (void**)&m_pGapStartRecorder_Device,  sizeof(short)*MSA_GA_POPULATION_SIZE*MSA_GA_CUDA_MAX_LEN );
	CUDA_MALLOC( (void**)&m_pGapPos_Tx_Device,  sizeof(short)*MSA_GA_POPULATION_SIZE );
}

void     CMSAGA_CUDA_Algorithm::ClearDevice()
{
	CUDA_FREE((void*)m_pPopulationDevice[m_curPopulationIndex]);
	CUDA_FREE((void*)m_pPopulationDevice[1-m_curPopulationIndex]);
	CUDA_FREE((void*)m_pOrganLenDevice[m_curOrganLenIndex]);
	CUDA_FREE((void*)m_pOrganLenDevice[1-m_curOrganLenIndex]);

	hiprandDestroyGenerator( m_RandomNumberGen );

	CUDA_FREE((void*)m_pSubMatDevice);

	//FITNESS
	//unit score
	CUDA_FREE((void*)m_pUnitScore_Device);
	//column score
	CUDA_FREE((void*)m_pColumnScore_Device);
	//organ score
	CUDA_FREE((void*)m_pOrganScore_Device);
	//weight
	CUDA_FREE((void*)m_pSeqWeight_Device);

	//SELECTION 
	CUDA_FREE((void*)m_pOrganScore_Align_Device );
	CUDA_FREE((void*)m_pOrganIndex_Align_Device );
	CUDA_FREE((void*)m_pOrganScore_Align_Sorted_Device );
	CUDA_FREE((void*)m_pOrganIndex_Align_Sorted_Device );
	CUDA_FREE((void*)m_pRandomOrganIdx_Device );
	CUDA_FREE((void*)m_pSelectedOrganIdx_Device );

	//MUTATION
	CUDA_FREE((void*)m_pMutation_RandOrgan_Device );
	CUDA_FREE((void*)m_pMutation_RandTx_Device );
	CUDA_FREE((void*)m_pMutation_RandTy_Device );
	CUDA_FREE((void*)m_pGapStartRecorder_Device );
	CUDA_FREE((void*)m_pGapPos_Tx_Device );

	//RECOMB
	CUDA_FREE((void*)m_pRecomb_RandOrgan_Device );
	CUDA_FREE((void*)m_pRecomb_RandTy_Device );

	CUDA_FREE((void*)m_pRecomb_RandOrganIdx0_Device );
	CUDA_FREE((void*)m_pRecomb_RandOrganIdx1_Device );

	//for vertical recombination
	CUDA_FREE((void*)m_pPos1_Min_Device );
	CUDA_FREE((void*)m_pPos1_Max_Device );

	CUDA_FREE((void*)m_pRecomb_RandTx_Device );

	CUDA_FREE((void*)m_pOrganIndex0_Device );
	CUDA_FREE((void*)m_pOrganIndex1_Device );
	CUDA_FREE((void*)m_pVerticalRecomb_pos0_Device );
	CUDA_FREE((void*)m_pVerticalRecomb_pos1_Device );

	//ALIGNMENT
	CUDA_FREE((void*)m_pGapRecorder_Device );
	CUDA_FREE((void*)m_pRangeRecorder_Device );
	CUDA_FREE((void*)m_pGapLenRecorder_Device );
}

void		CMSAGA_CUDA_Algorithm::InitDevice()
{
	//init the random number generator
	hiprandCreateGenerator( &m_RandomNumberGen,  HIPRAND_RNG_PSEUDO_XORWOW );

	//set the substitution matrix.
	SetSubstitutionMat2GPU( MSAGA_MAT_TYPE );
	InitFitness();
	InitSelection();
	InitMutation();
	InitRecomb();
	InitArrangement();

	//�������ĳ�ʼ����
	dim3 thread(MSA_GA_CUDA_BLOCK_SIZE, MSA_GA_CUDA_BLOCK_SIZE);
	dim3 grid(ceil((float)MSA_GA_POPULATION_SIZE / (thread.x*thread.y)), 1);
	MSAGA_CALC_SEQ_LEN_UNLIMIT<32> << < grid, thread >> >(false, m_pOrganLenDevice[1 - m_curOrganLenIndex], m_pOrganLenDevice[m_curOrganLenIndex],
														  m_pPopulationDevice[m_curPopulationIndex], m_pRandomOrganIdx_Device, 1,
														  MSA_GA_CUDA_MAX_LEN, grid.x*thread.x, MSA_GA_POPULATION_SIZE);
	m_curOrganLenIndex = 1-m_curOrganLenIndex;
}

void		CMSAGA_CUDA_Algorithm::RandomNumberGeneratorDevice( float *pRandNum, int nRand )
{
	hiprandStatus_t  curandResult = hiprandGenerateUniform( m_RandomNumberGen, pRandNum, nRand );
    if (curandResult != HIPRAND_STATUS_SUCCESS)
    {
		throw   CAppException( DEF_EXCEPTION_UNEXPECTED,DEF_EXCEPTION_LEVEL_EXIT_TRY_BLOCK
			,__EXCEPTION_SITE__ ,"Could not generate random numbers! ");
    }
}

void		CMSAGA_CUDA_Algorithm::SelectionDeviceUnlimit()
{
	//align the organ score and index with  power-of-two array lengths
	//the non used score initialized with -1e4;
	dim3 thread(m_nOrganSize_Align, 1);
	dim3 grid(1, 1);
	if (m_nOrganSize_Align >= SHARED_SIZE_LIMIT)
	{
		thread.x = MSA_GA_CUDA_BLOCK_SIZE;
		thread.y = MSA_GA_CUDA_BLOCK_SIZE;
		grid.x = ceil((float)m_nOrganSize_Align / (MSA_GA_CUDA_BLOCK_SIZE*MSA_GA_CUDA_BLOCK_SIZE));
	}
	//dbgWriteFile("OrganScore_Org.txt", m_pOrganScore_Device, MSA_GA_POPULATION_SIZE, MSA_GA_POPULATION_SIZE);
	MSAGA_ALIGN_CUDA_UNLIMIT<32> << < grid, thread >> >(m_pOrganScore_Align_Device, m_pOrganIndex_Align_Device, m_pOrganScore_Device, MSA_GA_POPULATION_SIZE, grid.x*thread.x);
	
	//sort the score using bitonic sort method
#if 0
	thread.x = SHARED_SIZE_LIMIT/2;
	thread.y = 1;
	grid.x =1;
	grid.y = 1;
    bitonicSortShared<<< grid, thread >>>(m_pOrganScore_Align_Sorted_Device, m_pOrganIndex_Align_Sorted_Device, 
		                                  m_pOrganScore_Align_Device, m_pOrganIndex_Align_Device, 
										  SHARED_SIZE_LIMIT, 0 );
#endif
	//dbgWriteFile("score_align.txt", m_pOrganScore_Align_Device, m_nOrganSize_Align, m_nOrganSize_Align);
	if (m_nOrganSize_Align < SHARED_SIZE_LIMIT)
	{
		bitonicSortShared(m_pOrganScore_Align_Sorted_Device, m_pOrganIndex_Align_Sorted_Device,
						  m_pOrganScore_Align_Device, m_pOrganIndex_Align_Device, 1, m_nOrganSize_Align, 0);
	}
	else
	{
		bitonicSort(m_pOrganScore_Align_Sorted_Device, m_pOrganIndex_Align_Sorted_Device,
			m_pOrganScore_Align_Device, m_pOrganIndex_Align_Device, 1, m_nOrganSize_Align, 0);
	}
	//dbgWriteFile("aligned_sorted_score.txt", m_pOrganScore_Align_Sorted_Device, m_nOrganSize_Align, m_nOrganSize_Align);

	CUDA_DEV2HOST(&m_bestScore, m_pOrganScore_Align_Sorted_Device, sizeof(float));

	//using tournament selection method to select the organ
	// Generate random numbers
	RandomNumberGeneratorDevice( m_pRandomOrganIdx_Device, MSA_GA_CUDA_SELECTION_CANDIDATE*MSA_GA_POPULATION_SIZE );

	//select the max one of the candidate
#if 0
	thread.x = MSA_GA_POPULATION_SIZE;
	thread.y = 1;
	grid.x = 1;
	grid.y = 1;
	MSAGA_SELECT_MAX_CUDA<32><<< grid, thread>>>(m_pSelectedOrganIdx_Device, m_pOrganIndex_Align_Sorted_Device, m_pOrganScore_Align_Sorted_Device, 
		                                         m_pRandomOrganIdx_Device, MSA_GA_POPULATION_SIZE, MSA_GA_CUDA_SELECTION_CANDIDATE );
#endif
	thread.x = MSA_GA_CUDA_BLOCK_SIZE;
	thread.y = thread.x;
	grid.x = ceil((float)MSA_GA_POPULATION_SIZE / (thread.x*thread.y));
	grid.y = 1;
	MSAGA_SELECT_MAX_CUDA_UNLIMIT <<<grid, thread >>>(m_pSelectedOrganIdx_Device, m_pOrganIndex_Align_Sorted_Device, m_pOrganScore_Align_Sorted_Device,
														m_pRandomOrganIdx_Device, MSA_GA_POPULATION_SIZE, MSA_GA_CUDA_SELECTION_CANDIDATE, grid.x*thread.x);
	//dbgWriteFile("m_pSelectedOrganIdx_Device.txt", m_pSelectedOrganIdx_Device, MSA_GA_POPULATION_SIZE, MSA_GA_POPULATION_SIZE);
	//dbgWriteSeqsFromDevice2File("before next gen.txt");
	//produce the next generation of organs.
	thread.x=MSA_GA_CUDA_BLOCK_SIZE;
	thread.y=thread.x;
	grid.x = ceil((float)m_nCellNum / (thread.x*thread.y));
	grid.y = 1;
	MSAGA_SELECT_NEXT_GEN_UNLIMIT<<< grid, thread>>>( m_pPopulationDevice[1-m_curPopulationIndex], m_pPopulationDevice[m_curPopulationIndex], 
													 m_pSelectedOrganIdx_Device, MSA_GA_CUDA_MAX_LEN, R_MSA_GA_CUDA_MAX_LEN,
													 POPULATION_WIDTH, R_POPULATION_WIDTH, grid.x*thread.x, m_nCellNum);
	m_curPopulationIndex = 1-m_curPopulationIndex;
	
//	dbgWriteSeqsFromDevice2File("after next gen.txt");

	//dbgCheckSequence();

	//record the next generation length
	thread.x = MSA_GA_CUDA_BLOCK_SIZE;
	thread.y = thread.x;
	grid.x = ceil((float)MSA_GA_POPULATION_SIZE / (thread.x*thread.y));
	grid.y = 1;
	MSAGA_RECORD_NEXT_GEN_LEN_UNLIMIT << <grid, thread >> >(m_pOrganLenDevice[1 - m_curOrganLenIndex], m_pOrganLenDevice[m_curOrganLenIndex], m_pSelectedOrganIdx_Device, grid.x*thread.x, MSA_GA_POPULATION_SIZE);
	m_curOrganLenIndex = 1 - m_curOrganLenIndex;
	//dbgWriteFile<unsigned int>("m_pOrganLen.txt", m_pOrganLenDevice[m_curOrganLenIndex], MSA_GA_POPULATION_SIZE, MSA_GA_POPULATION_SIZE);
	//dbgWriteSeqsFromDevice2File("checkPass.txt");
}

void		CMSAGA_CUDA_Algorithm::RecombinationDeviceUnlimit()
{
	RecombinationHorizentalDeviceUnlimit();
	//dbgCheckSequence();
	RecombinationVerticalDeviceUnlimit();
}

void		CMSAGA_CUDA_Algorithm::RecombinationDevice()
{
	//dbgWriteSeqsFromDevice2File( "seqResult0.txt" );
	RecombinationHorizentalDevice();
	//dbgWriteSeqsFromDevice2File( "seqResult0.txt" );
    RecombinationVerticalDevice();
	//dbgWriteSeqsFromDevice2File( "seqResult1.txt" );
	//static int count=0;
	//++count;
	//if(count==2)
	//{
	//	int i=0;
	//}
}

void		CMSAGA_CUDA_Algorithm::RecombinationVerticalDeviceUnlimit()
{
	int	nRecomb = (MSA_GA_POPULATION_SIZE-1)/2;
	RandomNumberGeneratorDevice( m_pRecomb_RandOrgan_Device, nRecomb );
	RandomNumberGeneratorDevice( m_pRecomb_RandTx_Device, nRecomb );
	RandomNumberGeneratorDevice( m_pRecomb_RandOrganIdx0_Device, nRecomb );
	RandomNumberGeneratorDevice( m_pRecomb_RandOrganIdx1_Device, nRecomb );

	//find the recomb pos of org0 and org1.
#if 0
	dim3 thread(nRecomb, 1);
	dim3 grid(1, 1 );
	MSAGA_RECOMB_VERTICAL_RECOMB_POS0_UNLIMIT<<< grid, thread>>>( m_pVerticalRecomb_pos0_Device, m_pOrganIndex0_Device, m_pOrganIndex1_Device,
																  m_pRecomb_RandOrgan_Device, m_pRecomb_RandTx_Device, 
																  m_pRecomb_RandOrganIdx0_Device, m_pRecomb_RandOrganIdx1_Device,
																  MSA_GA_VERTICAL_RECOMB_RATIO, m_pOrganLenDevice[m_curOrganLenIndex], thread.x*grid.x);
#endif 
	dim3 thread(MSA_GA_CUDA_BLOCK_SIZE, MSA_GA_CUDA_BLOCK_SIZE);
	dim3 grid(ceil((float)nRecomb/(thread.x*thread.y)), 1);
	MSAGA_RECOMB_VERTICAL_RECOMB_POS0_UNLIMIT << < grid, thread >> >(m_pVerticalRecomb_pos0_Device, m_pOrganIndex0_Device, m_pOrganIndex1_Device,
																	 m_pRecomb_RandOrgan_Device, m_pRecomb_RandTx_Device,
																	 m_pRecomb_RandOrganIdx0_Device, m_pRecomb_RandOrganIdx1_Device,
																	 MSA_GA_VERTICAL_RECOMB_RATIO, m_pOrganLenDevice[m_curOrganLenIndex], thread.x*grid.x, nRecomb, MSA_GA_POPULATION_SIZE);

	grid.x = ceil((float)nRecomb*m_pPopulation->nSeqSize / (thread.x*thread.y));
	MSAGA_RECOMB_VERTICAL_RECOMB_POS1_UNLIMIT<<< grid, thread >>>(m_pVerticalRecomb_pos0_Device, m_pVerticalRecomb_pos1_Device, m_pOrganIndex0_Device, m_pOrganIndex1_Device,
																	m_pRecomb_RandOrgan_Device, MSA_GA_VERTICAL_RECOMB_RATIO, thread.x*grid.x, 
																	m_pPopulationDevice[m_curPopulationIndex], POPULATION_WIDTH, MSA_GA_CUDA_MAX_LEN, nRecomb, 1.0f / nRecomb,
																	CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE), m_pPopulation->nSeqSize);

	//dbgWriteFile("m_pVerticalRecomb_pos0_Device.txt", m_pVerticalRecomb_pos0_Device, nRecomb, nRecomb);
	//dbgWriteFile("m_pVerticalRecomb_pos1_Device.txt", m_pVerticalRecomb_pos1_Device, nRecomb*m_pPopulation->nSeqSize, nRecomb);
	//dbgWriteFile("m_pOrganIndex0_Device.txt", m_pOrganIndex0_Device, nRecomb, nRecomb);
	//dbgWriteFile("m_pOrganIndex1_Device.txt", m_pOrganIndex1_Device, nRecomb, nRecomb);

	grid.x = ceil((float)nRecomb/(thread.x*thread.y));
	//find the min and max value of org1 recomb position 
	MSAGA_RECOMB_VERTICAL_POS1_MIN_MAX_UNLIMIT<32><<< grid, thread>>>( m_pPos1_Min_Device, m_pPos1_Max_Device, m_pVerticalRecomb_pos1_Device, m_pPopulation->nSeqSize,
															           MSA_GA_VERTICAL_RECOMB_RATIO, m_pRecomb_RandOrgan_Device, grid.x*thread.x, nRecomb );

	hipMemset( (void*)m_pPopulationDevice[1-m_curPopulationIndex], MSA_GA_CUDA_SPACE, m_nCellNum*sizeof(short) );
	grid.x = ceil((float)m_nCellNum/(thread.x*thread.y));
	MSAGA_RECOMB_VERTICAL_UNLIMIT<<< grid, thread>>>( m_pPopulationDevice[1-m_curPopulationIndex], m_pPopulationDevice[m_curPopulationIndex],  
													  m_pOrganIndex0_Device, m_pOrganIndex1_Device, m_pVerticalRecomb_pos0_Device, m_pVerticalRecomb_pos1_Device,													
													  m_pRecomb_RandOrgan_Device, MSA_GA_VERTICAL_RECOMB_RATIO, m_pPos1_Min_Device, m_pPos1_Max_Device,														
													  CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE), MSA_GA_CUDA_MAX_LEN, R_MSA_GA_CUDA_MAX_LEN, 
													  POPULATION_WIDTH, R_POPULATION_WIDTH, MSA_GA_POPULATION_SIZE, thread.x*grid.x, m_nCellNum);
	m_curPopulationIndex = 1-m_curPopulationIndex;
	//dbgCheckSequence();
	ArrangeSeqDeviceUnlimit( m_pRecomb_RandOrgan_Device, MSA_GA_VERTICAL_RECOMB_RATIO, true );
}

void		CMSAGA_CUDA_Algorithm::RecombinationVerticalDevice()
{
	int	nRecomb = (MSA_GA_POPULATION_SIZE-1)/2;
	RandomNumberGeneratorDevice( m_pRecomb_RandOrgan_Device, nRecomb );
	RandomNumberGeneratorDevice( m_pRecomb_RandTx_Device, nRecomb );
	RandomNumberGeneratorDevice( m_pRecomb_RandOrganIdx0_Device, nRecomb );
	RandomNumberGeneratorDevice( m_pRecomb_RandOrganIdx1_Device, nRecomb );

	dim3 thread;
	dim3 grid;
	if( nRecomb*m_pPopulation->nSeqSize>1024 )
	{
		thread.x = 1024/m_pPopulation->nSeqSize;
		thread.y = m_pPopulation->nSeqSize;
		grid.x = ceil((float)nRecomb*m_pPopulation->nSeqSize/1024);
		grid.y = 1;
	}
	else
	{
		thread.x = nRecomb;
		thread.y = m_pPopulation->nSeqSize;
		grid.x = 1;
		grid.y = 1;
	}
	//find the recomb pos of org0 and org1.
	MSAGA_RECOMB_VERTICAL_RECOMB_POS<32><<< grid, thread>>>( m_pVerticalRecomb_pos0_Device, m_pVerticalRecomb_pos1_Device, m_pOrganIndex0_Device, m_pOrganIndex1_Device,
																														  m_pPopulationDevice[m_curPopulationIndex], m_pRecomb_RandOrgan_Device, m_pRecomb_RandTx_Device,
																														  m_pRecomb_RandOrganIdx0_Device, m_pRecomb_RandOrganIdx1_Device,
																														  MSA_GA_VERTICAL_RECOMB_RATIO, MSA_GA_POPULATION_SIZE, MSA_GA_CUDA_MAX_LEN,
																														  CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE), m_pOrganLenDevice[m_curOrganLenIndex], grid.x*thread.x, nRecomb );

	//dbgWriteFile( std::string("Recomb_Organ0.txt"), m_pOrganIndex0_Device, nRecomb, nRecomb );
	//dbgWriteFile( std::string("Recomb_Organ1.txt"), m_pOrganIndex1_Device, nRecomb, nRecomb );
	//dbgWriteFile( std::string("Recomb_pos0.txt"), m_pVerticalRecomb_pos0_Device, nRecomb*m_pPopulation->nSeqSize, nRecomb );
	//dbgWriteFile( std::string("Recomb_pos1.txt"), m_pVerticalRecomb_pos1_Device, nRecomb*m_pPopulation->nSeqSize, nRecomb );
	
	thread.x = nRecomb;
	thread.y = 1;
	grid.x = 1;
	grid.y = 1;
	//find the min and max value of org1 recomb position 
	MSAGA_RECOMB_VERTICAL_POS1_MIN_MAX<32><<< grid, thread>>>( m_pPos1_Min_Device, m_pPos1_Max_Device, m_pVerticalRecomb_pos1_Device, m_pPopulation->nSeqSize,
		                                                                                                                          MSA_GA_VERTICAL_RECOMB_RATIO, m_pRecomb_RandOrgan_Device,  nRecomb );

	//writeDebugFile( std::string("m_pPos1_Min_Device.txt"), m_pPos1_Min_Device, nRecomb );
	//writeDebugFile( std::string("m_pPos1_Max_Device.txt"), m_pPos1_Max_Device, nRecomb );

	//dbgWriteSeqsFromDevice2File( "seqResult0.txt" );
	thread.x = MSA_GA_CUDA_MAX_LEN;
	thread.y = m_pPopulation->nSeqSize;
	grid.x = MSA_GA_POPULATION_SIZE;
	grid.y = 1;
	MSAGA_RECOMB_VERTICAL<32><<< grid, thread>>>( m_pPopulationDevice[1-m_curPopulationIndex], m_pPopulationDevice[m_curPopulationIndex],  
		                                                                                         m_pOrganIndex0_Device, m_pOrganIndex1_Device, m_pVerticalRecomb_pos0_Device, m_pVerticalRecomb_pos1_Device,
		                                                                                         m_pRecomb_RandOrgan_Device, MSA_GA_VERTICAL_RECOMB_RATIO, 
																								 MSA_GA_POPULATION_SIZE*MSA_GA_CUDA_MAX_LEN, MSA_GA_POPULATION_SIZE, m_pPopulation->nSeqSize, 
																								 m_pPos1_Min_Device, m_pPos1_Max_Device,
																								 CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE), MSA_GA_CUDA_SPACE );
	m_curPopulationIndex = 1-m_curPopulationIndex;
	
	//dbgWriteSeqsFromDevice2File( "seqResult1.txt" );
	//writeDebugFile( std::string("BX_VERTICAL.txt"), m_pVerticalRecomb_pos1_Device, nRecomb );
	//dbgWriteSeqsFromDevice2File( "seqResult1.txt" );

	//arrange the organs.
	ArrangeSeqDevice( m_pRecomb_RandOrgan_Device, 1 );
}


void		CMSAGA_CUDA_Algorithm::RecombinationHorizentalDeviceUnlimit()
{
	int	nRecomb = (MSA_GA_POPULATION_SIZE-1)/2;
	RandomNumberGeneratorDevice( m_pRecomb_RandOrgan_Device, nRecomb );
	RandomNumberGeneratorDevice( m_pRecomb_RandTy_Device, nRecomb );
	RandomNumberGeneratorDevice( m_pRecomb_RandOrganIdx0_Device, nRecomb );
	RandomNumberGeneratorDevice( m_pRecomb_RandOrganIdx1_Device, nRecomb );

	dim3 thread(MSA_GA_CUDA_BLOCK_SIZE, MSA_GA_CUDA_BLOCK_SIZE );
	dim3 grid(ceil((float)m_nCellNum/(thread.x*thread.y)), 1);
	MSAGA_RECOMB_HORIZEN_UNLIMIT<<< grid, thread>>>( m_pPopulationDevice[1-m_curPopulationIndex], m_pPopulationDevice[m_curPopulationIndex],
													 m_pRecomb_RandOrgan_Device,m_pRecomb_RandTy_Device, m_pRecomb_RandOrganIdx0_Device, 
													 m_pRecomb_RandOrganIdx1_Device, MSA_GA_HORIZENTAL_RECOMB_RATIO, 
													 MSA_GA_CUDA_MAX_LEN, R_MSA_GA_CUDA_MAX_LEN, POPULATION_WIDTH, R_POPULATION_WIDTH,
													 thread.x*grid.x, MSA_GA_POPULATION_SIZE, m_OrganismSize, m_nCellNum);
	m_curPopulationIndex = 1-m_curPopulationIndex;
	//dbgCheckSequence();
	ArrangeSeqDeviceUnlimit( m_pRecomb_RandOrgan_Device, MSA_GA_HORIZENTAL_RECOMB_RATIO, true );
}

void		CMSAGA_CUDA_Algorithm::RecombinationHorizentalDevice()
{
	int	nRecomb = (MSA_GA_POPULATION_SIZE-1)/2;
	RandomNumberGeneratorDevice( m_pRecomb_RandOrgan_Device, nRecomb );
	RandomNumberGeneratorDevice( m_pRecomb_RandTy_Device, nRecomb );
	RandomNumberGeneratorDevice( m_pRecomb_RandOrganIdx0_Device, nRecomb );
	RandomNumberGeneratorDevice( m_pRecomb_RandOrganIdx1_Device, nRecomb );

	dim3 thread(MSA_GA_CUDA_MAX_LEN, m_pPopulation->nSeqSize );
	dim3 grid(MSA_GA_POPULATION_SIZE, 1 );
	MSAGA_RECOMB_HORIZEN<32><<< grid, thread>>>( m_pPopulationDevice[1-m_curPopulationIndex], m_pPopulationDevice[m_curPopulationIndex],
	                                                                                             m_pRecomb_RandOrgan_Device,m_pRecomb_RandTy_Device, m_pRecomb_RandOrganIdx0_Device, m_pRecomb_RandOrganIdx1_Device,
																								 MSA_GA_HORIZENTAL_RECOMB_RATIO, m_pPopulation->nSeqSize,  MSA_GA_POPULATION_SIZE );
	m_curPopulationIndex = 1-m_curPopulationIndex;

	//arrange the organs.
	ArrangeSeqDevice( m_pRecomb_RandOrgan_Device, MSA_GA_HORIZENTAL_RECOMB_RATIO );
}

void		CMSAGA_CUDA_Algorithm::MutationDeviceUnlimit()
{
	MutationGapInsertDeviceUnlimit();
	MutationGapExtensionDeviceUnlimit();
	MutationGapReductionDeviceUnlimit();
}

void		CMSAGA_CUDA_Algorithm::MutationDevice()
{
   //dbgWriteSeqsFromDevice2File( "seqResult0.txt" );
	MutationGapInsertDevice();
	MutationGapExtensionDevice();
	MutationGapReductionDevice();
	//dbgWriteSeqsFromDevice2File( "seqResult1.txt" );
}

void		CMSAGA_CUDA_Algorithm::Evolution()
{
	//The evolution main code with cuda
	//dbgCheckSequence();
	SelectionDeviceUnlimit();
	//dbgCheckSequence();
	MutationDeviceUnlimit();
	//dbgCheckSequence();
	RecombinationDeviceUnlimit();
	//dbgCheckSequence();
}

void     CMSAGA_CUDA_Algorithm::testSPScoreCPU( const COrganism& vOrgan )
{
	//calculate col score
	// Allocate host memory for organs
	unsigned int size_colscore = m_pPopulation->nSeqSize*MSA_GA_CUDA_MAX_LEN;
	unsigned int mem_size_colscore = sizeof(short) * size_colscore;
	short *colscore = (short *)malloc(mem_size_colscore);
	memset(colscore, 0, mem_size_colscore);

	for( int i=0; i<vOrgan.pSequence[0].sequence.getLen(); ++i )
	{
		for( int j=0; j<vOrgan.nSeqSize-1; ++j )
		{
			short score=0;
			for( int k=j+1; k<vOrgan.nSeqSize; ++k )
			{
				int a =  vOrgan.pSequence[j].sequence.getSequenceContext().at(i).m_iCode;
				int b =  vOrgan.pSequence[k].sequence.getSequenceContext().at(i).m_iCode;
				if ( a != CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE) && b != CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE) )
				{
					score += CGlobalSpace::m_sAlignParams.m_SubstitutionMatMgr.getSubMatrixScore( MSAGA_MAT_TYPE, a, b );
				}
			}
			colscore[ j*MSA_GA_CUDA_MAX_LEN+i ] = score;
		}
	}
	std::auto_ptr<SeqAnsis::CFileWriter>		pFileWriter( new SeqAnsis::CFileWriter( std::string("seqColScore.txt") ) );
	pFileWriter->OutputVector( colscore,  size_colscore);
}

void		CMSAGA_CUDA_Algorithm::testColSumScoreCPU( const COrganism& vOrgan )
{
	//calculate col score
	// Allocate host memory for organs
	unsigned int size_colscore = m_pPopulation->nSeqSize*MSA_GA_CUDA_MAX_LEN;
	unsigned int mem_size_colscore = sizeof(short) * size_colscore;
	short *colscore = (short *)malloc(mem_size_colscore);
	memset(colscore, 0, mem_size_colscore);

	unsigned int size_colsumscore = MSA_GA_CUDA_MAX_LEN;
	unsigned int mem_size_colsumscore = sizeof(short) * size_colsumscore;
	short *colsumscore = (short *)malloc(mem_size_colsumscore);
	memset(colsumscore, 0, mem_size_colsumscore);

	for( int i=0; i<vOrgan.pSequence[0].sequence.getLen(); ++i )
	{
		for( int j=0; j<vOrgan.nSeqSize-1; ++j )
		{
			short score=0;
			for( int k=j+1; k<vOrgan.nSeqSize; ++k )
			{
				int a =  vOrgan.pSequence[j].sequence.getSequenceContext().at(i).m_iCode;
				int b =  vOrgan.pSequence[k].sequence.getSequenceContext().at(i).m_iCode;
				if ( a != CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE) && b != CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE) )
				{
					score += CGlobalSpace::m_sAlignParams.m_SubstitutionMatMgr.getSubMatrixScore( MSAGA_MAT_TYPE, a, b );
				}
			}
			colscore[ j*MSA_GA_CUDA_MAX_LEN+i ] = score;
		}
	}

	for( int i=0; i<vOrgan.pSequence[0].sequence.getLen(); ++i )
	{
		for( int j=0; j<vOrgan.nSeqSize; ++j )
		{
			colsumscore[i] += colscore[ j*MSA_GA_CUDA_MAX_LEN+i ];
		}
	}

	std::auto_ptr<SeqAnsis::CFileWriter>		pFileWriter( new SeqAnsis::CFileWriter( std::string("seqColSumScore.txt") ) );
	pFileWriter->OutputVector( colsumscore,  size_colsumscore);
}

int		CMSAGA_CUDA_Algorithm::SPScore(const std::vector<CSequence>& vInputSequences)
{
	int spScore = 0;
	//pair score
	for (int j = 0; j<vInputSequences[0].getLen(); ++j)
	{
		for (int k = 0; k<vInputSequences.size() - 1; ++k)
		{
			for (int m = k + 1; m<vInputSequences.size(); ++m)
			{
				int a = vInputSequences[k].getSequenceContext().at(j).m_iCode;
				int b = vInputSequences[m].getSequenceContext().at(j).m_iCode;
				if (a != CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE) && b != CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE))
				{
					spScore += CGlobalSpace::m_sAlignParams.m_SubstitutionMatMgr.getSubMatrixScore(MSAGA_MAT_TYPE, a, b);
				}
			}
		}
	}
	//gap penalty
	for (int j = 0; j<vInputSequences.size(); ++j)
	{
		int gapNum = 0;
		int idx = 0;
		while (idx<vInputSequences[j].getLen())
		{
			while (idx<vInputSequences[j].getLen() && vInputSequences[j].getSequenceContext().at(idx).m_iCode != CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE))	++idx;
			while (idx<vInputSequences[j].getLen() && vInputSequences[j].getSequenceContext().at(idx).m_iCode == CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE))
			{
				++gapNum;
				++idx;
			}
			if (gapNum>0)
			{
				spScore += CGlobalSpace::m_sAlignParams.m_SubstitutionMatMgr.getGapOpenCost(MSAGA_MAT_TYPE) + (gapNum - 1)*CGlobalSpace::m_sAlignParams.m_SubstitutionMatMgr.getGapExtendCost(MSAGA_MAT_TYPE);
			}
			gapNum = 0;
		}
	}
	return spScore;
}

int		CMSAGA_CUDA_Algorithm::SPScore( const COrganism& vOrgan )
{
	int spScore = 0;
	//pair score
	for ( int j=0; j<vOrgan.pSequence[0].sequence.getLen(); ++j )
	{
		for ( int k=0; k<vOrgan.nSeqSize-1; ++k )
		{
			for ( int m=k+1; m<vOrgan.nSeqSize; ++m )
			{
				int a =  vOrgan.pSequence[k].sequence.getSequenceContext().at(j).m_iCode;
				int b =  vOrgan.pSequence[m].sequence.getSequenceContext().at(j).m_iCode;
				if ( a != CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE) && b != CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE) )
				{
					spScore += CGlobalSpace::m_sAlignParams.m_SubstitutionMatMgr.getSubMatrixScore( MSAGA_MAT_TYPE, a, b );
				}
			}
		}
	}
	//gap penalty
	for ( int j=0; j<vOrgan.nSeqSize; ++j )
	{
		int gapNum=0;
		int idx=0;
		while( idx<vOrgan.pSequence[j].sequence.getLen() )
		{
			while( idx<vOrgan.pSequence[j].sequence.getLen() && vOrgan.pSequence[j].sequence.getSequenceContext().at(idx).m_iCode != CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE) )	++idx;
			while( idx<vOrgan.pSequence[j].sequence.getLen() && vOrgan.pSequence[j].sequence.getSequenceContext().at(idx).m_iCode == CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE) )	
			{
				++gapNum;
				++idx;
			}
			if( gapNum>0 )
			{
				spScore += CGlobalSpace::m_sAlignParams.m_SubstitutionMatMgr.getGapOpenCost(MSAGA_MAT_TYPE) + (gapNum-1)*CGlobalSpace::m_sAlignParams.m_SubstitutionMatMgr.getGapExtendCost(MSAGA_MAT_TYPE);
			}
			gapNum=0;
		}
	}
	return spScore;
}

void		CMSAGA_CUDA_Algorithm::ArrangeSeqDevice(float*	pRandOrgan_Device, float randLimit)
{

}

void		CMSAGA_CUDA_Algorithm::ArrangeSeqDeviceUnlimit( float*	pRandOrgan_Device, float randLimit,	bool bRecomb )
{
	//��Ҫ���пո�ȥ������һ������洢�ռ�ȫ����Ϊδʹ��
	hipMemset( (void*)m_pPopulationDevice[1-m_curPopulationIndex], MSA_GA_CUDA_SPACE, m_nCellNum*sizeof(short) );

	//�ڽ϶�����β�����ո񣬲��볤��
	dim3 thread(MSA_GA_CUDA_BLOCK_SIZE, MSA_GA_CUDA_BLOCK_SIZE);
	dim3 grid( ceil( (float)m_nCellNum/(thread.x*thread.y) ), 1 );
	MSAGA_FILL_SPACE_AT_TAIL_UNLIMIT<<< grid, thread>>>( bRecomb, m_pPopulationDevice[1-m_curPopulationIndex], m_pPopulationDevice[m_curPopulationIndex], 
														 pRandOrgan_Device, randLimit, thread.x*grid.x, 													
														 CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE), 													
														 MSA_GA_CUDA_MAX_LEN, R_MSA_GA_CUDA_MAX_LEN,
														 POPULATION_WIDTH, R_POPULATION_WIDTH,
														 m_pPopulation->nSeqSize, m_nCellNum, MSA_GA_POPULATION_SIZE);
	m_curPopulationIndex = 1-m_curPopulationIndex;
	//dbgCheckSequence();
	//��¼ȫΪ�ո����	
	thread.x=MSA_GA_CUDA_MAX_LEN;	
	thread.y=1;	
	grid.x=MSA_GA_POPULATION_SIZE;	
	MSAGA_GAP_RECORDER<32><<< grid, thread>>>( bRecomb, m_pGapRecorder_Device, m_pPopulationDevice[m_curPopulationIndex], pRandOrgan_Device, randLimit, 	
											   CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE),												 	
											   MSA_GA_POPULATION_SIZE*MSA_GA_CUDA_MAX_LEN, m_OrganismSize, MSA_GA_POPULATION_SIZE );													

	//ɾ��ȫ��Ϊ�ո����(������β���ո�)
	thread.x = MSA_GA_CUDA_BLOCK_SIZE;	
	thread.y = thread.x;	
	grid.x = ceil( (float)m_nCellNum/(thread.x*thread.y) );	
	grid.y = 1;		
	hipMemset( (void*)m_pPopulationDevice[1-m_curPopulationIndex], MSA_GA_CUDA_SPACE, m_nCellNum*sizeof(short) );
	MSAGA_DELETE_SPACE_UNLIMIT<<< grid, thread>>>( bRecomb, m_pPopulationDevice[1-m_curPopulationIndex], m_pPopulationDevice[m_curPopulationIndex],	
												   pRandOrgan_Device, randLimit, m_pGapRecorder_Device, thread.x*grid.x,
												   MSA_GA_POPULATION_SIZE, POPULATION_WIDTH, R_POPULATION_WIDTH, MSA_GA_CUDA_MAX_LEN, R_MSA_GA_CUDA_MAX_LEN, m_nCellNum);
	m_curPopulationIndex = 1-m_curPopulationIndex;	
	//dbgCheckSequence();
	//����ɾ��β���ո�
#if 0
	thread.x = MSA_GA_CUDA_BLOCK_SIZE;
	thread.y = thread.x;
	grid.x = ceil((float)MSA_GA_CUDA_MAX_LEN*m_pPopulation->nSeqSize*MSA_GA_POPULATION_SIZE / (thread.x*thread.y));
	grid.y = 1;

	hipMemset((void*)m_pPopulationDevice[1 - m_curPopulationIndex], MSA_GA_CUDA_SPACE, m_nCellNum*sizeof(short));
	MSAGA_DELETE_SPACE_AT_TAIL_UNLIMIT <<< grid, thread >>>(bRecomb, m_pPopulationDevice[1 - m_curPopulationIndex], m_pPopulationDevice[m_curPopulationIndex],
															pRandOrgan_Device, randLimit, thread.x*grid.x, POPULATION_WIDTH, R_POPULATION_WIDTH,
															MSA_GA_CUDA_MAX_LEN, R_MSA_GA_CUDA_MAX_LEN, m_pPopulation->nSeqSize, 
															CGlobalSpace::m_sAlignParams.getAminoAcidChar2IntCode(GENESPACE), m_nCellNum, MSA_GA_POPULATION_SIZE);
	m_curPopulationIndex = 1 - m_curPopulationIndex;
#endif
	//dbgCheckSequence();
	//���¼��������ĸ��峤��
#if 0
	thread.x = MSA_GA_CUDA_BLOCK_SIZE;
	thread.y = MSA_GA_CUDA_BLOCK_SIZE;
#endif
	grid.x = ceil((float)MSA_GA_POPULATION_SIZE / (thread.x*thread.y));
	grid.y=1;
	MSAGA_CALC_SEQ_LEN_UNLIMIT<32><<< grid, thread>>>( bRecomb, m_pOrganLenDevice[1-m_curOrganLenIndex], m_pOrganLenDevice[m_curOrganLenIndex], 
													   m_pPopulationDevice[m_curPopulationIndex], pRandOrgan_Device, randLimit, MSA_GA_CUDA_MAX_LEN, 
													   grid.x*thread.x, MSA_GA_POPULATION_SIZE);
	m_curOrganLenIndex = 1-m_curOrganLenIndex;
}

//==================FOR CPU VERIFICATION PURPOSE=====================//
void		CMSAGA_CUDA_Algorithm::Mutation()
{
	GapInsertMutation();
	//GapExtensionMutation();
	//GapReductionMutation();
}

void		CMSAGA_CUDA_Algorithm::GapInsertMutation()
{
	for ( int i=1; i<MSA_GA_POPULATION_SIZE; ++i )
	{
		if ( CGlobalSpace::m_sUtility.getRandomNumber() < MSA_GA_BLOCK_MUTATION )
		{
				int iSeqNo = CGlobalSpace::m_sUtility.getRandomNumber(m_pPopulation[i].nSeqSize);
				//only one gap insert?
				int iBlockNum = CGlobalSpace::m_sUtility.getRandomNumber(MSA_GA_MAX_INSERT_GAP)+1;
				CSeqData& Seq = m_pPopulation[i].pSequence[iSeqNo].sequence.getSequenceContext();				
				int idxSelectedPos = CGlobalSpace::m_sUtility.getRandomNumber(m_pPopulation[i].pSequence[iSeqNo].sequence.getLen()-1);
				CSeqData::iterator&	itrSeqData = Seq.begin();
				for ( int j=0; j<idxSelectedPos; ++j )
				{
					++itrSeqData;
				}
				if ( itrSeqData==Seq.end() )
				{
					throw CAppException( DEF_EXCEPTION_INDEX_OUT_OF_RANGE, DEF_EXCEPTION_LEVEL_EXIT_TRY_BLOCK, __EXCEPTION_SITE__, "false in the pointer!" );
				}
				//insert gaps
				for ( int j=0; j<iBlockNum; ++j )
				{
					itrSeqData = Seq.insert( itrSeqData, StruSeqElem(GENESPACE,-1) );
				}
				ResizeOrgan( m_pPopulation[i] );
			}
	}
}

int        CMSAGA_CUDA_Algorithm::getSeqLen( const CSequence& Seq )
{
	const CSeqData& SeqData = Seq.getSequenceContext();
	int seqLen = SeqData.size();
	int count=0;
	for( int i=0; i<seqLen; ++i )
	{
		if( SeqData[i].m_char != GENESPACE&&SeqData[i].m_char!=NONEGENE )
		{
			++count;
		}
	}
	return count;
}

void     CMSAGA_CUDA_Algorithm::dbgCheckSequence()
{
	unsigned int size_organs = m_pPopulation->nSeqSize*MSA_GA_CUDA_MAX_LEN*MSA_GA_POPULATION_SIZE;
	unsigned int mem_size_organs = sizeof(short) * size_organs;
	short *h_organs = (short *)malloc(mem_size_organs);
	//read sequences from device to host
	CUDA_DEV2HOST(   h_organs, m_pPopulationDevice[m_curPopulationIndex], mem_size_organs );

	//ÿһ�������е�ÿһ�����н��г���У��
	int nWidth = MSA_GA_CUDA_MAX_LEN*MSA_GA_POPULATION_SIZE;
	for( int k=0; k<MSA_GA_POPULATION_SIZE; ++k )
	{
		for( int i=0; i<m_pPopulation->nSeqSize; ++i )
		{
			int nSeqLen=0;
			for ( int j=0; j<MSA_GA_CUDA_MAX_LEN; ++j )
			{
				if( h_organs[ k*MSA_GA_CUDA_MAX_LEN+i*nWidth+j ] != MSA_GA_CUDA_SPACE )
				{
					char acid = CGlobalSpace::m_sAlignParams.getAminoAcidInt2CharCode( h_organs[k*MSA_GA_CUDA_MAX_LEN+i*nWidth+j ] );
					if( acid!='-' )
					{
						++nSeqLen;
					}
				}
				else
				{
					while( j<MSA_GA_CUDA_MAX_LEN && h_organs[ k*MSA_GA_CUDA_MAX_LEN+i*nWidth+j ] == MSA_GA_CUDA_SPACE )
					{
						++j;
					}
					if( j<MSA_GA_CUDA_MAX_LEN && h_organs[ k*MSA_GA_CUDA_MAX_LEN+i*nWidth+j ] != MSA_GA_CUDA_SPACE )
					{
						char ch[256];
						sprintf_s(ch,  "The pos %i of seq %i of the population %i is checksum error!" , j, i,  k);
						CGlobalSpace::m_sEventLog.writeEvent(ch);
						dbgWriteSeqsFromDevice2File( "checkError.txt" );
					}
				}
			}
			if( nSeqLen!=m_OrginSequences[i].getLen() )
			{
				char ch[256];
				sprintf_s(ch,  "The seq %i of the population %i is wrong, seq len in device=%i, org seq len=%i  The evolution run is %i" , i,  k, nSeqLen, m_OrginSequences[i].getLen(), m_iRun);
				CGlobalSpace::m_sEventLog.writeEvent(ch);
				dbgWriteSeqsFromDevice2File( "checkError.txt" );
			}
		}
	}

	free(h_organs);
	h_organs=NULL;

	//dbgWriteSeqsFromDevice2File( "checkPass.txt" );
}

void     CMSAGA_CUDA_Algorithm::dbgWriteSeqsFromDevice2File( const std::string& filename )
{
	std::fstream   fileOut;
	if( !fileOut.is_open() )
	{
		try
		{
			fileOut.open( filename.c_str(), std::ios::out|std::ios::trunc );
		}
		catch (CAppException const& e)
		{
			throw	e;
		}
	}
	unsigned int size_organs = m_pPopulation->nSeqSize*MSA_GA_CUDA_MAX_LEN*MSA_GA_POPULATION_SIZE;
	unsigned int mem_size_organs = sizeof(short) * size_organs;
	short *h_organs = (short *)malloc(mem_size_organs);
	//read sequences from device to host
	CUDA_DEV2HOST(   h_organs, m_pPopulationDevice[m_curPopulationIndex], mem_size_organs );

	//��ÿһ�������ӡ���ļ�
	int nWidth = MSA_GA_CUDA_MAX_LEN*MSA_GA_POPULATION_SIZE;
	for( int k=0; k<MSA_GA_POPULATION_SIZE; ++k )
	{
		fileOut<<"==========Population  "<<k<<"=========="<<std::endl;
		for( int i=0; i<m_pPopulation->nSeqSize; ++i )
		{
			for ( int j=0; j<MSA_GA_CUDA_MAX_LEN; ++j )
			{
				if( h_organs[ k*MSA_GA_CUDA_MAX_LEN+i*nWidth+j ] != MSA_GA_CUDA_SPACE )
				{
					char acid = CGlobalSpace::m_sAlignParams.getAminoAcidInt2CharCode( h_organs[k*MSA_GA_CUDA_MAX_LEN+i*nWidth+j ] );
					fileOut<<acid;
				}
				else
				{
					break;
				}
			}
			fileOut<<std::endl;//�������ͬһ���������һ������
		}
		fileOut<<std::endl<<std::endl;//��һ���������һ������
	}
	fileOut.close();
	free(h_organs);
	h_organs=NULL;
}

};
